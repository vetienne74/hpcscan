
//-------------------------------------------------------------------------------------------------------
// This class handles all queries related to the hardware
// Specialized class that targets NVIDIA GPUs
// Derived class from Hardware
// Associated to the test modes CUDA, CUDA_Ref & CUDA_Opt
//-------------------------------------------------------------------------------------------------------

#include "hardware_Cuda.h"

#include <nvml.h>

#include "config.h"
#include "constant.h"
#include "global.h"
#include "output_report.h"

using namespace std;

namespace hpcscan {

//-------------------------------------------------------------------------------------------------------

Hardware_Cuda::Hardware_Cuda(string gridMode) : Hardware(gridMode)
						{
	printDebug(MID_DEBUG, "IN Hardware_Cuda::Hardware_Cuda");

	// TODO

	printDebug(MID_DEBUG, "OUT Hardware_Cuda::Hardware_Cuda");
						}

//-------------------------------------------------------------------------------------------------------

Hardware_Cuda::~Hardware_Cuda(void)
{
	printDebug(MID_DEBUG, "IN Hardware_Cuda::~Hardware_Cuda");

	// TODO

	printDebug(MID_DEBUG, "OUT Hardware_Cuda::~Hardware_Cuda");
}

//-------------------------------------------------------------------------------------------------------

void Hardware_Cuda::info(void)
{
	printDebug(MID_DEBUG, "IN Hardware_Cuda::info");

	printInfo(MASTER, " Hardware information") ;
	printInfo(MASTER, " NVIDIA GPU") ;

	// display all available GPUs
	int startDevice = 0;
	int endDevice = 0;
	int deviceCount;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess) {
		printError(" In Grid_Cuda::info, hipGetDeviceCount", (int) error_id) ;
	}

	if (deviceCount == 0) {
		printError(" No GPU found") ;
	}
	else
	{
		printInfo(MASTER, " Number of GPUs found", deviceCount) ;
	}

	startDevice = 0;
	endDevice = deviceCount - 1;

	for (int currentDevice = startDevice; currentDevice <= endDevice;
			currentDevice++) {
		hipDeviceProp_t deviceProp;
		hipError_t error_id = hipGetDeviceProperties(&deviceProp, currentDevice);

		if (error_id == hipSuccess) {
			string deviceStr = " Device #" + to_string(currentDevice) + "\t";
			printInfo(MASTER, deviceStr, deviceProp.name) ;

			if (deviceProp.computeMode == hipComputeModeProhibited) {
				printError(" Error: device is running in <Compute Mode Prohibited>") ;
			}
		} else {
			printf("hipGetDeviceProperties returned %d\n-> %s\n", (int)error_id,
					hipGetErrorString(error_id));
		}
	}

	// CUDA aware library
	if (Config::Instance()->gpuMpiAware)
	{
		printInfo(MASTER, " MPI GPU-Aware Library", "ENABLED") ;
	}
	else
	{
		printInfo(MASTER, " MPI GPU-Aware Library", "DISABLED") ;
	}

	// support for power usage
	if (supportGetPowerUsage())
	{
		printInfo(MASTER, " Read power usage", "SUPPORTED") ;
	}
	else
	{
		printInfo(MASTER, " Read power usage", "NOT SUPPORTED") ;
	}

	print_line5() ;

	printDebug(MID_DEBUG, "OUT Hardware_Cuda::info");
}

//-------------------------------------------------------------------------------------------------------

bool Hardware_Cuda::supportGetPowerUsage(void)
{
	printDebug(MID_DEBUG, "IN Hardware_Cuda::supportGetPowerUsage");

	bool retVal = false ;

	// get power consumption
	{
		//nvmlDeviceGetPowerUsage (nvmlDevice_t device, unsigned int* power)
		unsigned int power , i ;
		nvmlReturn_t result;

		// First initialize NVML library
		result = nvmlInit();
		if (NVML_SUCCESS != result)
		{
			printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
		}

		nvmlDevice_t device;
		i = 0 ;
		result = nvmlDeviceGetHandleByIndex(i, &device);
		if (NVML_SUCCESS != result)
		{
			printf("Failed to get handle for device %u: %s\n", i, nvmlErrorString(result));
		}

		result = nvmlDeviceGetPowerUsage(device, &power) ;
		if (result != NVML_SUCCESS)
		{
			printInfo(MASTER, " nvmlDeviceGetPowerUsage", nvmlErrorString(result)) ;
		}
		else
		{
			printInfo(MASTER, " nvmlDeviceGetPowerUsage (mWatt)", (Myint) power) ;
			retVal = true ;
		}
	}

	printDebug(MID_DEBUG, "OUT Hardware_Cuda::supportGetPowerUsage");
	return(retVal) ;
}

} // namespace hpcscan
