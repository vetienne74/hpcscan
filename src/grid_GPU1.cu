#include "hip/hip_runtime.h"

//-------------------------------------------------------------------------------------------------------
// Derived class from Grid
// Optimized for GPU
// Version 1 ??
//-------------------------------------------------------------------------------------------------------

#include "grid_GPU1.h"

#include <algorithm> // for min and max
#include <cassert>
#include <cfloat>  // for FLT_MAX ;
#include <cmath>   // for fabs
#include <cstddef> // for NULL
#include <fstream>
#include <stdio.h>

#include "mpi.h"

#include "config.h"
#include "constant.h"
#include "fdm.h"
#include "global.h"
#include "output_report.h"

#include <thrust/device_vector.h>
#include <thrust/reduce.h>

using namespace std;

namespace hpcscan {

//-------------------------------------------------------------------------------------------------------

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
	hipError_t e=hipGetLastError();                                 \
	if(e!=hipSuccess) {                                              \
	  printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
	  exit(0); \
	}                                                                 \
   }

//-------------------------------------------------------------------------------------------------------

// there is probably an easier way to implement this (3d blocks?)
__global__ void cuda_fill_const(Myfloat *data, Myfloat val, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End)
{
	int size = n1*n2*n3;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	// 1D to 3D index (thanks StackOverflow)
	// public int[] to3D( int idx ) {
	// 	final int z = idx / (xMax * yMax);
	// 	idx -= (z * xMax * yMax);
	// 	final int y = idx / xMax;
	// 	final int x = idx % xMax;
	// 	return new int[]{ x, y, z };
	// }

	while (tid < size)
	{
		int t_i3 = tid / (n1*n2);
		int idx = tid-t_i3*n1*n2;
		int t_i2 = idx/n1;
		int t_i1 = idx%n1;

		if (t_i1 >= i1Start && t_i1 <= i1End &&
			t_i2 >= i2Start && t_i2 <= i2End &&
			t_i3 >= i3Start && t_i3 <= i3End   )
		{
			data[tid] = val;
			//printf("data[%d]=%f\n",tid,val);
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------

__global__ void cuda_fill_sine(Myfloat *data, Myfloat64 param1, Myfloat64 param2, Myfloat64 param3, Myfloat64 amp, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End, Myfloat Orig1, Myfloat Orig2, Myfloat Orig3, Myfloat64 d1, Myfloat64 d2, Myfloat64 d3 )
{
	// printf("sine %f %f %f %f %f\n",param1,param2,param3,amp,Orig1);

	int size = n1*n2*n3;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		int t_i3 = tid / (n1*n2);
		int idx = tid-t_i3*n1*n2;
		int t_i2 = idx/n1;
		int t_i1 = idx%n1;

		if (t_i1 >= i1Start && t_i1 <= i1End &&
			t_i2 >= i2Start && t_i2 <= i2End &&
			t_i3 >= i3Start && t_i3 <= i3End   )
		{
			Myfloat64 coord1 = Myfloat64(Orig1 + t_i1 * d1);
			Myfloat64 coord2 = Myfloat64(Orig2 + t_i2 * d2);
			Myfloat64 coord3 = Myfloat64(Orig3 + t_i3 * d3);

			Myfloat val = amp * sin(coord1 * param1) * sin(coord2 * param2) * sin(coord3 * param3);

			data[tid] = val;
			//printf("data[%d]=%f\n",tid,val);
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------

__global__ void cuda_fill_linear(Myfloat *data, Myfloat64 param1, Myfloat64 param2, Myfloat64 param3, Myfloat64 amp, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End, Myfloat Orig1, Myfloat Orig2, Myfloat Orig3, Myfloat64 d1, Myfloat64 d2, Myfloat64 d3)
{
	// printf("linear %f %f %f %f\n",param1,param2,param3,amp);

	int size = n1*n2*n3;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		int t_i3 = tid / (n1*n2);
		int idx = tid-t_i3*n1*n2;
		int t_i2 = idx/n1;
		int t_i1 = idx%n1;

		if (t_i1 >= i1Start && t_i1 <= i1End &&
			t_i2 >= i2Start && t_i2 <= i2End &&
			t_i3 >= i3Start && t_i3 <= i3End   )
		{
			Myfloat64 coord1 = Myfloat64(Orig1 + t_i1 * d1);
			Myfloat64 coord2 = Myfloat64(Orig2 + t_i2 * d2);
			Myfloat64 coord3 = Myfloat64(Orig3 + t_i3 * d3);

			Myfloat val = amp * coord1 * coord2 * coord3;

			data[tid] = val;
			//printf("data[%d]=%f\n",tid,val);
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------

//-------------------------------------------------------------------------------------------------------

__global__ void cuda_diff(Myfloat *data1, Myfloat *data2, Myfloat *dataOut, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End)
{
	int size = n1*n2*n3;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		int t_i3 = tid / (n1*n2);
		int idx = tid-t_i3*n1*n2;
		int t_i2 = idx/n1;
		int t_i1 = idx%n1;

		dataOut[tid]=0;

		if (t_i1 >= i1Start && t_i1 <= i1End &&
			t_i2 >= i2Start && t_i2 <= i2End &&
			t_i3 >= i3Start && t_i3 <= i3End   )
		{
			dataOut[tid] = fabsf(data1[tid]-data2[tid]);
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------

__global__ void cuda_fabsf(Myfloat *data, Myfloat *dataOut, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End)
{
	int size = n1*n2*n3;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		int t_i3 = tid / (n1*n2);
		int idx = tid-t_i3*n1*n2;
		int t_i2 = idx/n1;
		int t_i1 = idx%n1;

		dataOut[tid]=0;

		if (t_i1 >= i1Start && t_i1 <= i1End &&
			t_i2 >= i2Start && t_i2 <= i2End &&
			t_i3 >= i3Start && t_i3 <= i3End   )
		{
			dataOut[tid] = fabsf(data[tid]);
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------

__global__ void cuda_mask(Myfloat *data, Myfloat *dataOut, Myfloat val, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End)
{
	int size = n1*n2*n3;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		int t_i3 = tid / (n1*n2);
		int idx = tid-t_i3*n1*n2;
		int t_i2 = idx/n1;
		int t_i1 = idx%n1;

		dataOut[tid]=val;

		if (t_i1 >= i1Start && t_i1 <= i1End &&
			t_i2 >= i2Start && t_i2 <= i2End &&
			t_i3 >= i3Start && t_i3 <= i3End   )
		{
			dataOut[tid] = data[tid];
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------

// #pragma omp parallel for collapse(2)
// for (Myint64 i3 = i3Start; i3<= i3End; i3++)
// 	for (Myint64 i2 = i2Start; i2<= i2End; i2++)
// 		for (Myint64 i1 = i1Start; i1<= i1End; i1++)
// 			prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
// 					coef[i1+i2*n1+i3*n1*n2] * lapla[i1+i2*n1+i3*n1*n2] ;

__global__ void cuda_updatePressure(Myfloat *prn, Myfloat *prc, Myfloat *coef, Myfloat *lapla, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End)
{
	int size = n1*n2*n3;
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		int t_i3 = tid / (n1*n2);
		int idx = tid-t_i3*n1*n2;
		int t_i2 = idx/n1;
		int t_i1 = idx%n1;

		// dataOut[tid]=val;

		if (t_i1 >= i1Start && t_i1 <= i1End &&
			t_i2 >= i2Start && t_i2 <= i2End &&
			t_i3 >= i3Start && t_i3 <= i3End   )
		{
			prn[tid]=2.0*prc[tid]-prn[tid]+coef[tid]*lapla[tid];
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------




















Grid_GPU1::Grid_GPU1(Grid_type gridTypeIn) : Grid(gridTypeIn)
														{
	printDebug(MID_DEBUG, "IN Grid_GPU1::Grid_GPU1");

	gridMode = "GPU1" ;

	printDebug(MID_DEBUG, "OUT Grid_GPU1::Grid_GPU1");
														}

//-------------------------------------------------------------------------------------------------------

Grid_GPU1::Grid_GPU1(Grid_type gridTypeIn, Dim_type dimIn,
		Myint64 n1InnerIn, Myint64 n2InnerIn, Myint64 n3InnerIn) : Grid(gridTypeIn, dimIn,
				n1InnerIn, n2InnerIn, n3InnerIn)
{
	printDebug(MID_DEBUG, "IN Grid_GPU1::Grid_GPU1");

	gridMode = "GPU1" ;

	printDebug(MID_DEBUG, "OUT Grid_GPU1::Grid_GPU1");
}

//-------------------------------------------------------------------------------------------------------

Grid_GPU1::~Grid_GPU1(void)
{
	printDebug(MID_DEBUG, "IN Grid_GPU1::~Grid_GPU1");

	//delete[] grid_3d ;
	hipFree(d_grid_3d);
	hipFree(d_help_3d);
	cudaCheckError();

	printDebug(MID_DEBUG, "OUT Grid_GPU1::~Grid_GPU1");
}

//-------------------------------------------------------------------------------------------------------

void Grid_GPU1::info(void)
{
	printDebug(FULL_DEBUG, "IN Grid_GPU1::info");

	// parent class info
	Grid::info() ;

	// additional info
	// TO DO

	printDebug(FULL_DEBUG, "IN Grid_GPU1::info");
}


//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_GPU1::FD_LAPLACIAN(Point_type pointType, const Grid& Wgrid, Myint fdOrder)
{
	printDebug(MID_DEBUG, "IN Grid_GPU1::FD_LAPLACIAN");

	// TO DO
	Grid::FD_LAPLACIAN(pointType, Wgrid, fdOrder) ;

	printDebug(MID_DEBUG, "OUT Grid_GPU1::FD_LAPLACIAN");
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_GPU1::computePressureWithFD(Grid& prcGridIn, Grid& coefGridIn, Myint fdOrder)
{

	printDebug(FULL_DEBUG, "In Grid_GPU1::computePressureWithFD") ;

	// TO DO
	Grid::computePressureWithFD(prcGridIn, coefGridIn, fdOrder) ;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::computePressureWithFD") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

void Grid_GPU1::initializeGrid(void)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::initializeGrid") ;

	Grid::initializeGrid() ; // this sets up halos etc.
	printf("test n1=%d n2=%d n3=%d\n",n1,n2,n3);

	if (d_grid_3d == NULL)
	{
		hipMalloc( (void**)&d_grid_3d, n1*n2*n3*sizeof(Myfloat) );
		cudaCheckError();

		hipMalloc( (void**)&d_help_3d, n1*n2*n3*sizeof(Myfloat) );
		cudaCheckError();
	}
	printDebug(FULL_DEBUG, "Out Grid_GPU1::initializeGrid") ;
}

//-------------------------------------------------------------------------------------------------------
void Grid_GPU1::fill(Point_type pointType, Myfloat val)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::fill") ;

	// Grid::fill(pointType, val) ; // fill CPU memory (remove me)
	
	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	Grid::getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);
	cuda_fill_const<<<1024,128>>>(d_grid_3d,val,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	hipDeviceSynchronize();
	cudaCheckError();

	printDebug(FULL_DEBUG, "Out Grid_GPU1::fill") ;
}

//-------------------------------------------------------------------------------------------------------
void Grid_GPU1::fill(Point_type pointType, Func_type t1,  Func_type t2, Func_type t3,
		Myfloat64 param1, Myfloat64 param2, Myfloat64 param3, Myfloat64 amp)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::fill") ;

	Grid::fill(pointType, t1,  t2, t3, param1, param2, param3, amp) ;


	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	Grid::getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	// we only use sine and linear for now
	int ok = 1;
	if (t1 != FUNC_SINE && t1 != FUNC_LINEAR) ok = 0;
	if (t2 != FUNC_SINE && t2 != FUNC_LINEAR) ok = 0;
	if (t3 != FUNC_SINE && t3 != FUNC_LINEAR) ok = 0;
	if (!ok) printError("CUDA: only FUNC_SINE and FUNC_LINEAR implemented");

	if ((t1==t2 && t2==t3)==false) printError("CUDA: func has to be same in each dimension");

	if (t1 == FUNC_SINE) cuda_fill_sine<<<1024,128>>>  (d_grid_3d,param1,param2,param3,amp,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End,Orig1,Orig2,Orig3,d1,d2,d2);
	else                 cuda_fill_linear<<<1024,128>>>(d_grid_3d,param1,param2,param3,amp,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End,Orig1,Orig2,Orig3,d1,d2,d2);


	printDebug(FULL_DEBUG, "Out Grid_GPU1::fill") ;
}

//-------------------------------------------------------------------------------------------------------
Myfloat Grid_GPU1::getMin(Point_type pointType)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::getMin") ;

	// TO DO
	// return(Grid::getMin(pointType)) ;

	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	Grid::getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	cuda_mask<<<1024,256>>>(d_grid_3d,d_help_3d,999,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);

	thrust::device_ptr<Myfloat> d_help_3d_ptr = thrust::device_pointer_cast(d_help_3d);
	thrust::device_ptr<hpcscan::Myfloat> vptr = thrust::min_element(thrust::device, d_help_3d_ptr, d_help_3d_ptr + n1*n2*n3);
	float val = *vptr;
	// printf("val %f\n",val);
	return val;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::getMin") ;
}

//-------------------------------------------------------------------------------------------------------
Myfloat Grid_GPU1::getMax(Point_type pointType)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::getMax") ;

	// TO DO
	// return(Grid::getMax(pointType)) ;

	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	Grid::getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	// cuda_mask<<<1024,256>>>(d_grid_3d,d_help_3d,0,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);

	thrust::device_ptr<Myfloat> d_help_3d_ptr = thrust::device_pointer_cast(d_grid_3d);
	thrust::device_ptr<hpcscan::Myfloat> vptr = thrust::max_element(thrust::device, d_help_3d_ptr, d_help_3d_ptr + n1*n2*n3);
	float val = *vptr;
	// printf("val %f\n",val);
	return val;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::getMax") ;
}

//-------------------------------------------------------------------------------------------------------
Myfloat Grid_GPU1::L1Err(Point_type pointType, const Grid& gridIn) const
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::L1Err") ;

	// TO DO
	// return(Grid::L1Err(pointType, gridIn)) ;

	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	Grid::getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	thrust::device_ptr<Myfloat> d_help_3d_ptr;

	cuda_diff<<<1024,256>>>(d_grid_3d,gridIn.d_grid_3d,d_help_3d,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	d_help_3d_ptr = thrust::device_pointer_cast(d_help_3d);
	double totErr = thrust::reduce(thrust::device, d_help_3d_ptr, d_help_3d_ptr + n1*n2*n3);

	double totArr;
	if (false)
	{
		cuda_fabsf<<<1024,256>>>(gridIn.d_grid_3d,d_help_3d,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		totArr = thrust::reduce(thrust::device, d_help_3d_ptr, d_help_3d_ptr + n1*n2*n3);
	}
	else // assuming grid values are positive
	{
		d_help_3d_ptr = thrust::device_pointer_cast(gridIn.d_grid_3d);
		totArr = thrust::reduce(thrust::device, d_help_3d_ptr, d_help_3d_ptr + n1*n2*n3);
	}

	hipDeviceSynchronize();

	if (totArr < MAX_ERR_FLOAT) totArr = 1.0 * npoint ;

	return totErr/totArr;

	

	printDebug(FULL_DEBUG, "Out Grid_GPU1::L1Err") ;
}
//-------------------------------------------------------------------------------------------------------
Rtn_code Grid_GPU1::updatePressure(Point_type pointType, const Grid& prcGrid,
		const Grid& coefGrid, const Grid& laplaGrid)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::updatePressure") ;

	// TO DO
	// return(Grid::updatePressure(pointType, prcGrid, coefGrid, laplaGrid)) ;

	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	Grid::getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	cuda_updatePressure<<<1024,256>>>(d_grid_3d, prcGrid.d_grid_3d, coefGrid.d_grid_3d, laplaGrid.d_grid_3d, n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End);

	hipDeviceSynchronize();

	printDebug(FULL_DEBUG, "Out Grid_GPU1::updatePressure") ;
}
//-------------------------------------------------------------------------------------------------------
Rtn_code Grid_GPU1::applyBoundaryCondition(BoundCond_type boundCondType)
{
printDebug(FULL_DEBUG, "In Grid_GPU1::applyBoundaryCondition") ;

// TO DO
printf("BNDNDNND\n");
return(Grid::applyBoundaryCondition(boundCondType)) ;

// cuda_updatePressure<<<1024,256>>>(d_grid_3d, prcGrid.d_grid_3d, coefGrid.d_grid_3d, laplaGrid.d_grid_3d, n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End);

hipDeviceSynchronize();

printDebug(FULL_DEBUG, "Out Grid_GPU1::applyBoundaryCondition") ;
}
} // namespace hpcscan
