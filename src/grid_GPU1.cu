#include "hip/hip_runtime.h"

//-------------------------------------------------------------------------------------------------------
// Derived class from Grid
// Optimized for GPU
// Version 1 ??
//-------------------------------------------------------------------------------------------------------

#include "grid_GPU1.h"

#include <algorithm> // for min and max
#include <cassert>
#include <cfloat>  // for FLT_MAX ;
#include <cmath>   // for fabs
#include <cstddef> // for NULL
#include <fstream>
#include <stdio.h>

#include "mpi.h"

#include "config.h"
#include "constant.h"
#include "fdm.h"
#include "global.h"
#include "output_report.h"

using namespace std;

namespace hpcscan {

//-------------------------------------------------------------------------------------------------------

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
	hipError_t e=hipGetLastError();                                 \
	if(e!=hipSuccess) {                                              \
	  printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
	  exit(0); \
	}                                                                 \
   }

//-------------------------------------------------------------------------------------------------------

__global__ void print_from_gpu(void) 
{
    printf("Hello World! from thread [%d,%d] \
        From device\n", threadIdx.x,blockIdx.x);
}

//-------------------------------------------------------------------------------------------------------

// first (wrong) implementation of filling data(0:n1*n2*n3)=val
// TODO we shouldn't ignore pointType
// we could replace it with a thrust:: one-liner
__global__ void cuda_fill_const(Myfloat *data, Myfloat val, int n1, int n2, int n3, Myint64 i1Start, Myint64 i1End, Myint64 i2Start, Myint64 i2End, Myint64 i3Start, Myint64 i3End)
{
        Myint64 s1 = i1End - i1Start + 1;
        Myint64 s2 = i2End - i2Start + 1;
        Myint64 s3 = i3End - i3Start + 1;

        int size = n1*n2*n3;
        int size2= s1*s2*s3;

        printf("size1:%d\n",size);
        printf("size2:%d\n",size2);
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size) 
    {
		data[tid] = val;
		//printf("data[%d]=%f\n",tid,val);
		tid += blockDim.x * gridDim.x;
    }
}

//-------------------------------------------------------------------------------------------------------















Grid_GPU1::Grid_GPU1(Grid_type gridTypeIn) : Grid(gridTypeIn)
														{
	printDebug(MID_DEBUG, "IN Grid_GPU1::Grid_GPU1");

	gridMode = "GPU1" ;

	printDebug(MID_DEBUG, "OUT Grid_GPU1::Grid_GPU1");
														}

//-------------------------------------------------------------------------------------------------------

Grid_GPU1::Grid_GPU1(Grid_type gridTypeIn, Dim_type dimIn,
		Myint64 n1InnerIn, Myint64 n2InnerIn, Myint64 n3InnerIn) : Grid(gridTypeIn, dimIn,
				n1InnerIn, n2InnerIn, n3InnerIn)
{
	printDebug(MID_DEBUG, "IN Grid_GPU1::Grid_GPU1");

	gridMode = "GPU1" ;

	printDebug(MID_DEBUG, "OUT Grid_GPU1::Grid_GPU1");
}

//-------------------------------------------------------------------------------------------------------

Grid_GPU1::~Grid_GPU1(void)
{
	printDebug(MID_DEBUG, "IN Grid_GPU1::~Grid_GPU1");

	//delete[] grid_3d ;
	hipFree(d_grid_3d);
	cudaCheckError();

	printDebug(MID_DEBUG, "OUT Grid_GPU1::~Grid_GPU1");
}

//-------------------------------------------------------------------------------------------------------

void Grid_GPU1::info(void)
{
	printDebug(FULL_DEBUG, "IN Grid_GPU1::info");

	// parent class info
	Grid::info() ;

	// additional info
	// TO DO

	printDebug(FULL_DEBUG, "IN Grid_GPU1::info");
}


//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_GPU1::FD_LAPLACIAN(Point_type pType, const Grid& Wgrid, Myint fdOrder)
{
	printDebug(MID_DEBUG, "IN Grid_GPU1::FD_LAPLACIAN");

	// TO DO
	Grid::FD_LAPLACIAN(pType, Wgrid, fdOrder) ;

	printDebug(MID_DEBUG, "OUT Grid_GPU1::FD_LAPLACIAN");
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_GPU1::computePressureWithFD(Grid& prcGridIn, Grid& coefGridIn, Myint fdOrder)
{

	printDebug(FULL_DEBUG, "In Grid_GPU1::computePressureWithFD") ;

	// TO DO
	Grid::computePressureWithFD(prcGridIn, coefGridIn, fdOrder) ;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::computePressureWithFD") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

void Grid_GPU1::initializeGrid(void)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::initializeGrid") ;

	// TO DO
	print_from_gpu<<<4,1>>>();
	print_from_gpu<<<4,1>>>();
	hipDeviceSynchronize();

	cudaCheckError();

	printf("test\n");

	Grid::initializeGrid() ; // this sets up halos etc.
	printf("test n1=%d n2=%d n3=%d\n",n1,n2,n3);

	hipMalloc( (void**)&d_grid_3d, n1*n2*n3*sizeof(Myfloat) );
	cudaCheckError();

	printDebug(FULL_DEBUG, "Out Grid_GPU1::initializeGrid") ;
}

//-------------------------------------------------------------------------------------------------------
void Grid_GPU1::fill(Point_type pointType, Myfloat val)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::fill") ;

	Grid::fill(pointType, val) ; // fill CPU memory (remove me)
        //pointtype
        Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
        Grid::getGridIndex(INNER_POINTS, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);
        printf("test i1=%d i2=%d i3=%d\n",i1End,i2End,i3End);
        cuda_fill_const<<<512,64>>>(d_grid_3d,val,n1,n2,n3,i1Start, i1End, i2Start, i2End, i3Start, i3End);
	cudaCheckError();

	printDebug(FULL_DEBUG, "Out Grid_GPU1::fill") ;
}

//-------------------------------------------------------------------------------------------------------
void Grid_GPU1::fill(Point_type pType, Func_type t1,  Func_type t2, Func_type t3,
		Myfloat64 param1, Myfloat64 param2, Myfloat64 param3, Myfloat64 amp)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::fill") ;

	// TO DO
	Grid::fill(pType, t1,  t2, t3, param1, param2, param3, amp) ;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::fill") ;
}

//-------------------------------------------------------------------------------------------------------
Myfloat Grid_GPU1::getMin(Point_type pType)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::getMin") ;

	// TO DO
	return(Grid::getMin(pType)) ;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::getMin") ;
}

//-------------------------------------------------------------------------------------------------------
Myfloat Grid_GPU1::getMax(Point_type pType)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::getMax") ;

	// TO DO
	return(Grid::getMax(pType)) ;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::getMax") ;
}

//-------------------------------------------------------------------------------------------------------
Myfloat Grid_GPU1::L1Err(Point_type pointType, const Grid& gridIn) const
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::L1Err") ;

	// TO DO
	return(Grid::L1Err(pointType, gridIn)) ;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::L1Err") ;
}
//-------------------------------------------------------------------------------------------------------
Rtn_code Grid_GPU1::updatePressure(Point_type pType, const Grid& prcGrid,
		const Grid& coefGrid, const Grid& laplaGrid)
{
	printDebug(FULL_DEBUG, "In Grid_GPU1::updatePressure") ;

	// TO DO
	return(Grid::updatePressure(pType, prcGrid, coefGrid, laplaGrid)) ;

	printDebug(FULL_DEBUG, "Out Grid_GPU1::updatePressure") ;
}

} // namespace hpcscan
