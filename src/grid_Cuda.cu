#include "hip/hip_runtime.h"

//-------------------------------------------------------------------------------------------------------
// This grid is activated with command line option -testMode CUDA
// Derived class from Grid
// CUDA implementation (target GPU)
//-------------------------------------------------------------------------------------------------------

#include "grid_Cuda.h"

#include <algorithm> // for min and max
#include <cassert>
#include <cfloat>  // for FLT_MAX ;
#include <cmath>   // for fabs
#include <cstddef> // for NULL
#include <fstream>
#include <stdio.h>

#include "mpi.h"

#include "config.h"
#include "constant.h"
#include "fdm.h"
#include "global.h"
#include "output_report.h"

using namespace std;

namespace hpcscan {

//-------------------------------------------------------------------------------------------------------

// Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() { \
		hipError_t e=hipGetLastError(); \
		if(e!=hipSuccess) { \
			printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
			printError(" CUDA ERROR") ; \
			exit(0); \
		} \
}

//*******************************************************************************************************
// BEGINING OF CUDA KERNELS
//*******************************************************************************************************

//-------------------------------------------------------------------------------------------------------
// retrieve minimum value (1st step of global reduction)
// multi-block reduction on the input array dataIn
// each block finds its minimum and stores into the array dataOut at entry dataOut[blockIdx.x]

__global__ void kernel_multiBlk_minval(Myfloat *dataIn, Myfloat *dataOut,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	// dynamic shared memory
	extern __shared__ Myfloat sdata[];

	// set to max float value
	sdata[threadIdx.x] = +FLT_MAX ;

	// each thread find its minimum
	while (tid < size)
	{
		// convert 1d index to 3d indexes
		unsigned int i3 = tid / (n1*n2) ;
		unsigned int idx = tid-i3*n1*n2 ;
		unsigned int i2 = idx/n1 ;
		unsigned int i1 = idx - i2*n1 ;

		// check if point fall into target area
		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			// update min value
			Myfloat val = dataIn[tid];
			if (val < sdata[threadIdx.x]) sdata[threadIdx.x] = val;
		}

		tid += blockDim.x * gridDim.x;
	}

	__syncthreads();

	// find minimum between all threads
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			Myfloat val = sdata[threadIdx.x + s];
			if (val < sdata[threadIdx.x]) sdata[threadIdx.x] = val;
		}
		__syncthreads();
	}

	// write result for the block into global array
	if (threadIdx.x == 0) dataOut[blockIdx.x] = sdata[0];
}

//-------------------------------------------------------------------------------------------------------
// retrieve minimum value (2nd step of global reduction)
// single block reduction on the input array dataInOut
// the minimum is stored at first entry dataInOut[0]

__global__ void kernel_singleBlk_minval(Myfloat *dataInOut, const int dataInOutSize)
{
	int idx = threadIdx.x;
	for (int size = dataInOutSize/2; size>0; size  >>= 1) {
		if (idx<size)
			if (dataInOut[idx+size] < dataInOut[idx]) dataInOut[idx] = dataInOut[idx+size];
		__syncthreads();
	}
}

//-------------------------------------------------------------------------------------------------------
// retrieve maximum value (1st step of global reduction)
// multi-block reduction on the input array dataIn
// each block finds its maximum and stores into the array dataOut at entry dataOut[blockIdx.x]

__global__ void kernel_multiBlk_maxval(Myfloat *dataIn, Myfloat *dataOut,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	// dynamic shared memory
	extern __shared__ Myfloat sdata[];

	// set to min float value
	sdata[threadIdx.x] = -FLT_MAX ;

	// each thread find its minimum
	while (tid < size)
	{
		// convert 1d index to 3d indexes
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		// check if point fall into target area
		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			// update max value
			Myfloat val = dataIn[tid];
			if (val > sdata[threadIdx.x]) sdata[threadIdx.x] = val;
		}

		tid += blockDim.x * gridDim.x;
	}

	__syncthreads();

	// find maximum between all threads
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			Myfloat val = sdata[threadIdx.x + s];
			if (val > sdata[threadIdx.x]) sdata[threadIdx.x] = val;
		}
		__syncthreads();
	}

	// write result for the block into global array
	if (threadIdx.x == 0) dataOut[blockIdx.x] = sdata[0];
}

//-------------------------------------------------------------------------------------------------------
// retrieve maximum value (2nd step of global reduction)
// single block reduction on the input array dataInOut
// the maximum is stored at first entry dataInOut[0]

__global__ void kernel_singleBlk_maxval(Myfloat *dataInOut, const int dataInOutSize)
{
	int idx = threadIdx.x;
	for (int size = dataInOutSize/2; size>0; size/=2) {
		if (idx<size)
			if (dataInOut[idx+size] > dataInOut[idx]) dataInOut[idx] = dataInOut[idx+size];
		__syncthreads();
	}
}

//-------------------------------------------------------------------------------------------------------
// sum abs values (1st step of global reduction)
// multi-block reduction on the input array dataIn
// each block does the sum and stores into the array dataOut at entry dataOut[blockIdx.x]

__global__ void kernel_multiBlk_sumAbs(Myfloat *dataIn, Myfloat *dataOut,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	// dynamic shared memory
	extern __shared__ float sdata[];

	// set to zero
	sdata[threadIdx.x] = 0.0 ;

	// each thread sums
	while (tid < size)
	{
		// convert 1d index to 3d indexes
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		// check if point fall into target area
		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			// update sum
			Myfloat val = fabs(dataIn[tid]) ;
			sdata[threadIdx.x] += val ;
		}

		tid += blockDim.x * gridDim.x;
	}

	__syncthreads();

	// sum between all threads
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			Myfloat val = sdata[threadIdx.x + s];
			sdata[threadIdx.x] += val;
		}
		__syncthreads();
	}

	// write result for the block into global array
	if (threadIdx.x == 0) dataOut[blockIdx.x] = sdata[0];
}

//-------------------------------------------------------------------------------------------------------
// sum abs diff values between 2 grids (1st step of global reduction)
// multi-block reduction on the input arrays dataIn1 & dataIn2
// each block does the sum and stores into the array dataOut at entry dataOut[blockIdx.x]

__global__ void kernel_multiBlk_sumAbsDiff(Myfloat *dataIn1, Myfloat *dataIn2, Myfloat *dataOut,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	// dynamic shared memory
	extern __shared__ Myfloat sdata[];

	// set to zero
	sdata[threadIdx.x] = 0.0 ;

	// each thread sums
	while (tid < size)
	{
		// convert 1d index to 3d indexes
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		// check if point fall into target area
		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			// update sum
			Myfloat val = fabs(dataIn1[tid] - dataIn2[tid]) ;
			sdata[threadIdx.x] += val ;
		}

		tid += blockDim.x * gridDim.x;
	}

	__syncthreads();

	// sum between all threads
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			Myfloat val = sdata[threadIdx.x + s];
			sdata[threadIdx.x] += val;
		}
		__syncthreads();
	}

	// write result for the block into global array
	if (threadIdx.x == 0) dataOut[blockIdx.x] = sdata[0];
}

//-------------------------------------------------------------------------------------------------------
// sum abs values and abs diff between 2 grids (1st step of global reduction)
// multi-block reduction on the input arrays dataIn1 and dataIn2
// each block does the sum abs diff and stores into the array dataOut1 at entry dataOut1[blockIdx.x]
// each block does the sum abs and stores into the array dataOut2 at entry dataOut2[blockIdx.x]

__global__ void kernel_multiBlk_sumAbsAndAbsDiff(Myfloat *dataIn1, Myfloat *dataIn2, Myfloat *dataOut1, Myfloat *dataOut2,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	// dynamic shared memory
	extern __shared__ Myfloat sdata[];

	// split array into 2, 1st half for sum abs diff, 2nd half for sum abs
	Myfloat* sdata1 = &(sdata[0]) ;
	Myfloat* sdata2 = &(sdata[blockDim.x]) ;

	// set to zero
	sdata1[threadIdx.x] = 0.0 ;
	sdata2[threadIdx.x] = 0.0 ;

	// each thread sums
	while (tid < size)
	{
		// convert 1d index to 3d indexes
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		// check if point fall into target area
		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			// update sum
			sdata1[threadIdx.x] += fabs(dataIn1[tid] - dataIn2[tid]) ;
			sdata2[threadIdx.x] += fabs(dataIn2[tid]) ;
		}

		tid += blockDim.x * gridDim.x;
	}

	__syncthreads();

	// sum between all threads
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			sdata1[threadIdx.x] += sdata1[threadIdx.x + s] ;
			sdata2[threadIdx.x] += sdata2[threadIdx.x + s] ;
		}
		__syncthreads();
	}

	// write result for the block into global array
	if (threadIdx.x == 0)
	{
		dataOut1[blockIdx.x] = sdata1[0];
		dataOut2[blockIdx.x] = sdata2[0];
	}
}

//-------------------------------------------------------------------------------------------------------
// sum values (absolute values) (2nd step of global reduction)
// single block reduction on the input array dataInOut
// the maximum is stored at first entry dataInOut[0]

__global__ void kernel_singleBlk_sum(Myfloat *dataInOut, const Myint dataInOutSize)
{
	int idx = threadIdx.x;
	for (int size = dataInOutSize/2; size>0; size/=2) {
		if (idx<size)
			dataInOut[idx] += dataInOut[idx+size];
		__syncthreads();
	}
}

//-------------------------------------------------------------------------------------------------------
// max error between 2 grids (1st step of global reduction)
// multi-block reduction on the input arrays dataIn1 & dataIn2
// each block finds its maximum and stores into the array dataOut at entry dataOut[blockIdx.x]

__global__ void kernel_multiBlk_maxErr(Myfloat *dataIn1, Myfloat *dataIn2, Myfloat *dataOut,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	// dynamic shared memory
	extern __shared__ Myfloat sdata[];

	// set to min float value
	sdata[threadIdx.x] = -FLT_MAX ;

	// each thread sums
	while (tid < size)
	{
		// convert 1d index to 3d indexes
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		// check if point fall into target area
		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			// update max
			Myfloat err2 ;

			// prevent divide by 0
			if (fabs(dataIn2[tid]) < MAX_ERR_FLOAT)
			{
				err2 = fabs(dataIn1[tid] - dataIn2[tid]) ;
			}
			else
			{
				err2 = fabs(dataIn1[tid] - dataIn2[tid]) / fabs(dataIn2[tid]) ;
			}

			if (err2 > sdata[threadIdx.x])
			{
				sdata[threadIdx.x] = err2 ;
			}
		}

		tid += blockDim.x * gridDim.x;
	}

	__syncthreads();

	// find maximum between all threads
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			Myfloat val = sdata[threadIdx.x + s];
			if (val > sdata[threadIdx.x]) sdata[threadIdx.x] = val;
		}
		__syncthreads();
	}

	// write result for the block into global array
	if (threadIdx.x == 0) dataOut[blockIdx.x] = sdata[0];
}

//-------------------------------------------------------------------------------------------------------
// fill grid with constant values

__global__ void kernel_fill_const(Myfloat *dataOut, const Myfloat val,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		// convert 1d index to 3d indexes
		unsigned int i3 = tid / (n1*n2) ;
		unsigned int idx = tid-i3*n1*n2 ;
		unsigned int i2 = idx/n1 ;
		unsigned int i1 = idx - i2*n1 ;

		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			dataOut[tid] = val;
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// fill grid with predefined functions val1, val2, val3, val4
//
__global__ void kernel_fill_function(Myfloat *dataOut, Myfloat64 *val1, Myfloat64 *val2, Myfloat64 *val3, const Myfloat64 val4,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End   )
		{
			dataOut[tid] =  val4 * val1[i1-i1Start] * val2[i2-i2Start] * val3[i3-i3Start];
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// update array prn (input/output)
// input arrays prc, coef, lapla

__global__ void kernel_updatePressure(Myfloat *prn, Myfloat *prc, Myfloat *coef, Myfloat *lapla,
		const Myint n1, const Myint n2, const int n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	const Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		if (i1 >= i1Start && i1 <= i1End &&
				i2 >= i2Start && i2 <= i2End &&
				i3 >= i3Start && i3 <= i3End)
		{
			prn[tid] = TWO * prc[tid] - prn[tid] + coef[tid] * lapla[tid];
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// perform boundaray condition
// copy grid inner points in to halos and revert sign of values

__global__ void kernel_applyBoundaryCondition(Dim_type dim, Myfloat *data,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint I1HALO1_neigh, const Myint64 i1halo1_i1Start, const Myint64 i1halo1_i1End, const Myint64 i1halo1_i2Start, const Myint64 i1halo1_i2End, const Myint64 i1halo1_i3Start, const Myint64 i1halo1_i3End,
		const Myint I1HALO2_neigh, const Myint64 i1halo2_i1Start, const Myint64 i1halo2_i1End, const Myint64 i1halo2_i2Start, const Myint64 i1halo2_i2End, const Myint64 i1halo2_i3Start, const Myint64 i1halo2_i3End,
		const Myint I2HALO1_neigh, const Myint64 i2halo1_i1Start, const Myint64 i2halo1_i1End, const Myint64 i2halo1_i2Start, const Myint64 i2halo1_i2End, const Myint64 i2halo1_i3Start, const Myint64 i2halo1_i3End,
		const Myint I2HALO2_neigh, const Myint64 i2halo2_i1Start, const Myint64 i2halo2_i1End, const Myint64 i2halo2_i2Start, const Myint64 i2halo2_i2End, const Myint64 i2halo2_i3Start, const Myint64 i2halo2_i3End,
		const Myint I3HALO1_neigh, const Myint64 i3halo1_i1Start, const Myint64 i3halo1_i1End, const Myint64 i3halo1_i2Start, const Myint64 i3halo1_i2End, const Myint64 i3halo1_i3Start, const Myint64 i3halo1_i3End,
		const Myint I3HALO2_neigh, const Myint64 i3halo2_i1Start, const Myint64 i3halo2_i1End, const Myint64 i3halo2_i2Start, const Myint64 i3halo2_i2End, const Myint64 i3halo2_i3Start, const Myint64 i3halo2_i3End)

{
	Myint64 size = n1*n2*n3;
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;

	while (tid < size)
	{
		unsigned int i3 = tid / (n1*n2);
		unsigned int idx = tid-i3*n1*n2;
		unsigned int i2 = idx/n1;
		unsigned int i1 = idx - i2*n1 ;

		// I1HALO1
		if (I1HALO1_neigh == MPI_PROC_NULL)
		{
			Myint64 iInner = i1halo1_i1End+1;
			if (tid == iInner+i2*n1+i3*n1*n2) data[tid] = 0.0 ;

			if (i1 >= i1halo1_i1Start && i1 <= i1halo1_i1End &&
					i2 >= i1halo1_i2Start && i2 <= i1halo1_i2End &&
					i3 >= i1halo1_i3Start && i3 <= i1halo1_i3End   )
			{
				data[tid] = -data[(iInner+iInner-i1)+i2*n1+i3*n1*n2];
			}
		}

		// I1HALO2
		if (I1HALO2_neigh == MPI_PROC_NULL)
		{
			Myint64 iInner = i1halo2_i1Start-1;
			if (tid == iInner+i2*n1+i3*n1*n2) data[tid] = 0.0 ;

			if (i1 >= i1halo2_i1Start && i1 <= i1halo2_i1End &&
					i2 >= i1halo2_i2Start && i2 <= i1halo2_i2End &&
					i3 >= i1halo2_i3Start && i3 <= i1halo2_i3End   )
			{
				data[tid] = -data[(iInner-(i1-iInner))+i2*n1+i3*n1*n2];
			}
		}

		if (dim >= DIM2)
		{
			// I2HALO1
			if (I2HALO1_neigh == MPI_PROC_NULL)
			{
				Myint64 iInner = i2halo1_i2End+1;
				if (tid == i1+iInner*n1+i3*n1*n2) data[tid] = 0.0 ;

				if (i1 >= i2halo1_i1Start && i1 <= i2halo1_i1End &&
						i2 >= i2halo1_i2Start && i2 <= i2halo1_i2End &&
						i3 >= i2halo1_i3Start && i3 <= i2halo1_i3End   )
				{
					data[tid] = -data[i1+(iInner+iInner-i2)*n1+i3*n1*n2];
				}
			}

			// I2HALO2
			if (I2HALO2_neigh == MPI_PROC_NULL)
			{
				Myint64 iInner = i2halo2_i2Start-1;
				if (tid == i1+iInner*n1+i3*n1*n2) data[tid] = 0.0 ;

				if (i1 >= i2halo2_i1Start && i1 <= i2halo2_i1End &&
						i2 >= i2halo2_i2Start && i2 <= i2halo2_i2End &&
						i3 >= i2halo2_i3Start && i3 <= i2halo2_i3End   )
				{
					data[tid] = -data[i1+(iInner-(i2-iInner))*n1+i3*n1*n2];
				}
			}
		}

		if (dim >= DIM3)
		{
			// I3HALO1
			if (I3HALO1_neigh == MPI_PROC_NULL)
			{
				Myint64 iInner = i3halo1_i3End+1;
				if (tid == i1+i2*n1+iInner*n1*n2) data[tid] = 0.0 ;

				if (i1 >= i3halo1_i1Start && i1 <= i3halo1_i1End &&
						i2 >= i3halo1_i2Start && i2 <= i3halo1_i2End &&
						i3 >= i3halo1_i3Start && i3 <= i3halo1_i3End   )
				{
					data[tid] = -data[i1+i2*n1+(iInner+iInner-i3)*n1*n2];
				}
			}

			// I3HALO2
			if (I3HALO2_neigh == MPI_PROC_NULL)
			{
				Myint64 iInner = i3halo2_i3Start-1;
				if (tid == i1+i2*n1+iInner*n1*n2) data[tid] = 0.0 ;

				if (i1 >= i3halo2_i1Start && i1 <= i3halo2_i1End &&
						i2 >= i3halo2_i2Start && i2 <= i3halo2_i2End &&
						i3 >= i3halo2_i3Start && i3 <= i3halo2_i3End   )
				{
					data[tid] = -data[i1+i2*n1+(iInner-(i3-iInner))*n1*n2];
				}
			}
		}

		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// update pressure wavefield (used in propagator) - 1D
// input/output prn
// input prc

__global__ void kernel_computePressureWithFD_1D_O2(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O2_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_computePressureWithFD_1D_O4(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O4_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_computePressureWithFD_1D_O6(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O6_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_computePressureWithFD_1D_O8(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O8_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_computePressureWithFD_1D_O10(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O10_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_computePressureWithFD_1D_O12(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O12_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_computePressureWithFD_1D_O14(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O14_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_computePressureWithFD_1D_O16(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				FD_D2_O16_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// update pressure wavefield (used in propagator) - 2D
// input/output prn
// input prc

__global__ void kernel_computePressureWithFD_2D_O2(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O2_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O2_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_2D_O4(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O4_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O4_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_2D_O6(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O6_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O6_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_2D_O8(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O8_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O8_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_2D_O10(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O10_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O10_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_2D_O12(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O12_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O12_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_2D_O14(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O14_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O14_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_2D_O16(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O16_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O16_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// update pressure wavefield (used in propagator) - 3D
// input/output prn
// input prc

__global__ void kernel_computePressureWithFD_3D_O2(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O2_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O2_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O2_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_3D_O4(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O4_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O4_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O4_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_3D_O6(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O6_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O6_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O6_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_3D_O8(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O8_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O8_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O8_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_3D_O10(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O10_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O10_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O10_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_3D_O12(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O12_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O12_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O12_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_3D_O14(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O14_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O14_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O14_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

__global__ void kernel_computePressureWithFD_3D_O16(const Dim_type dim, const Myint fdOrder, Myfloat *prn, Myfloat *prc, Myfloat *coef,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		prn[i1+i2*n1+i3*n1*n2] = TWO * prc[i1+i2*n1+i3*n1*n2] - prn[i1+i2*n1+i3*n1*n2] +
				coef[i1+i2*n1+i3*n1*n2] *
				(FD_D2_O16_N1(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O16_N2(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
						+ FD_D2_O16_N3(prc, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// compute derivative along axis 1
// input u
// output w

__global__ void kernel_FD_D2_N1_O2(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O2_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N1_O4(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O4_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N1_O6(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O6_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N1_O8(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O8_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N1_O10(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O10_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N1_O12(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O12_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N1_O14(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O14_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N1_O16(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O16_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// compute derivative along axis 2
// input u
// output w

__global__ void kernel_FD_D2_N2_O2(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O2_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N2_O4(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O4_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N2_O6(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O6_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N2_O8(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O8_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N2_O10(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O10_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N2_O12(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O12_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N2_O14(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O14_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N2_O16(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O16_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// compute derivative along axis 3
// input u
// output w

__global__ void kernel_FD_D2_N3_O2(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O2_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N3_O4(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O4_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N3_O6(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O6_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N3_O8(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O8_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N3_O10(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O10_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N3_O12(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O12_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N3_O14(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O14_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_D2_N3_O16(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O16_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// compute 2D Laplacian
// input u
// output w

__global__ void kernel_FD_LAPLACIAN_2D_O2(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O2_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O2_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_2D_O4(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O4_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O4_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_2D_O6(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O6_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O6_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_2D_O8(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O8_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O8_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_2D_O10(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O10_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O10_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_2D_O12(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O12_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O12_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_2D_O14(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O14_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O14_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_2D_O16(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O16_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O16_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// compute 3D Laplacian
// input u
// output w

__global__ void kernel_FD_LAPLACIAN_3D_O2(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O2_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O2_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O2_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_3D_O4(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O4_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O4_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O4_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_3D_O6(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O6_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O6_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O6_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_3D_O8(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O8_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O8_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O8_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_3D_O10(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O10_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O10_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O10_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_3D_O12(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O12_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O12_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O12_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_3D_O14(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O14_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O14_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O14_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

__global__ void kernel_FD_LAPLACIAN_3D_O16(const Myint fdOrder, Myfloat *w, Myfloat *u,
		const Myfloat inv2_d1, const Myfloat inv2_d2, const Myfloat inv2_d3,
		const Myint n1, const Myint n2, const Myint n3,
		const Myint64 i1Start, const Myint64 i1End, const Myint64 i2Start, const Myint64 i2End, const Myint64 i3Start, const Myint64 i3End)
{
	unsigned int i1 = threadIdx.x + blockIdx.x * blockDim.x ;
	unsigned int i2 = threadIdx.y + blockIdx.y * blockDim.y ;
	unsigned int i3 = threadIdx.z + blockIdx.z * blockDim.z ;

	if (i1 >= i1Start && i1 <= i1End &&
			i2 >= i2Start && i2 <= i2End &&
			i3 >= i3Start && i3 <= i3End   )
	{
		w[i1+i2*n1+i3*n1*n2] =
				FD_D2_O16_N1(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O16_N2(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3)
				+ FD_D2_O16_N3(u, i1, i2, i3, inv2_d1, inv2_d2, inv2_d3, n1, n2, n3) ;
	}
}

//-------------------------------------------------------------------------------------------------------
// fill gridOut with val

__global__ void kernel_fillArray(Myfloat *gridOut, const Myfloat val, const Myint64 gridSize)
{
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < gridSize)
	{
		gridOut[tid] = val ;
		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// copy gridIn into gridOut

__global__ void kernel_copyArray(Myfloat *gridOut, Myfloat *gridIn, const Myint64 gridSize)
{
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < gridSize)
	{
		gridOut[tid] = gridIn[tid] ;
		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// sum gridIn1 and gridIn2 and stores into gridOut

__global__ void kernel_addArray(Myfloat *gridOut, Myfloat *gridIn1, Myfloat *gridIn2, const Myint64 gridSize)
{
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < gridSize)
	{
		gridOut[tid] = gridIn1[tid] + gridIn2[tid] ;
		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// multiply gridIn1 and gridIn2 and stores into gridOut

__global__ void kernel_multiplyArray(Myfloat *gridOut, Myfloat *gridIn1, Myfloat *gridIn2, const Myint64 gridSize)
{
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < gridSize)
	{
		gridOut[tid] = gridIn1[tid] * gridIn2[tid] ;
		tid += blockDim.x * gridDim.x;
	}
}

//-------------------------------------------------------------------------------------------------------
// add gridIn and gridOut and stores into gridOut

__global__ void kernel_addUpdateArray(Myfloat *gridOut, Myfloat *gridIn, const Myint64 gridSize)
{
	Myint64 tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < gridSize)
	{
		gridOut[tid] = gridOut[tid] + gridIn[tid] ;
		tid += blockDim.x * gridDim.x;
	}
}

//*******************************************************************************************************
// END OF CUDA KERNELS
//*******************************************************************************************************

//-------------------------------------------------------------------------------------------------------

Grid_Cuda::Grid_Cuda(Grid_type gridTypeIn) : Grid(gridTypeIn)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::Grid_Cuda");

	gridMode = GRID_MODE_CUDA ;

	d_grid_3d   = NULL ;
	d_help_3d   = NULL ;
	d_help_3d_2 = NULL ;

	// for kernels using 1D blocks
	gpuBlkSize  = Config::Instance()->gpuBlkSize ;
	gpuGridSize = Config::Instance()->gpuGridSize ;

	// for kernels using 3D blocks
	gpuBlkSize1  = Config::Instance()->gpuBlkSize1 ;
	gpuBlkSize2  = Config::Instance()->gpuBlkSize2 ;
	gpuBlkSize3  = Config::Instance()->gpuBlkSize3 ;
	gpuGridSize1 = 0 ;
	gpuGridSize2 = 0 ;
	gpuGridSize3 = 0 ;

	printDebug(MID_DEBUG, "OUT Grid_Cuda::Grid_Cuda");
}

//-------------------------------------------------------------------------------------------------------

Grid_Cuda::Grid_Cuda(Grid_type gridTypeIn, Dim_type dimIn,
		Myint64 n1InnerIn, Myint64 n2InnerIn, Myint64 n3InnerIn) : Grid(gridTypeIn, dimIn,
				n1InnerIn, n2InnerIn, n3InnerIn)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::Grid_Cuda");

	gridMode = GRID_MODE_CUDA ;

	d_grid_3d   = NULL ;
	d_help_3d   = NULL ;
	d_help_3d_2 = NULL ;

	// for kernels using 1D blocks
	gpuBlkSize  = Config::Instance()->gpuBlkSize ;
	gpuGridSize = Config::Instance()->gpuGridSize ;

	// for kernels using 3D blocks
	gpuBlkSize1  = Config::Instance()->gpuBlkSize1 ;
	gpuBlkSize2  = Config::Instance()->gpuBlkSize2 ;
	gpuBlkSize3  = Config::Instance()->gpuBlkSize3 ;
	gpuGridSize1 = 0 ;
	gpuGridSize2 = 0 ;
	gpuGridSize3 = 0 ;

	printDebug(MID_DEBUG, "OUT Grid_Cuda::Grid_Cuda");
}

//-------------------------------------------------------------------------------------------------------

Grid_Cuda::~Grid_Cuda(void)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::~Grid_Cuda");

	//delete[] grid_3d ;
	hipFree(d_grid_3d);
	hipFree(d_help_3d);
	hipFree(d_help_3d_2);
	cudaCheckError();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::~Grid_Cuda");
}


//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::info(void)
{
	printDebug(FULL_DEBUG, "IN Grid_Cuda::info");

	// parent class info
	Grid::info() ;

	// additional info
	printInfo(MASTER, "") ;
	printInfo(MASTER, " * GPU parameters * ") ;
	printInfo(MASTER, " - Kernel with 1D blocks") ;
	printInfo(MASTER, " Blocks per grid", gpuGridSize) ;
	printInfo(MASTER, " Threads per block", gpuBlkSize) ;
	printInfo(MASTER, " - Kernel with 3D blocks") ;
	printInfo(MASTER, " Blocks/grid axis1", gpuGridSize1) ;
	printInfo(MASTER, " Blocks/grid axis2", gpuGridSize2) ;
	printInfo(MASTER, " Blocks/grid axis3", gpuGridSize3) ;
	printInfo(MASTER, " Threads/block axis1", gpuBlkSize1) ;
	printInfo(MASTER, " Threads/block axis2", gpuBlkSize2) ;
	printInfo(MASTER, " Threads/block axis3", gpuBlkSize3) ;

	if (Config::Instance()->gpuMpiAware)
	{
		printInfo(MASTER, " MPI GPU-Aware Library", "ENABLED") ;
	}
	else
	{
		printInfo(MASTER, " MPI GPU-Aware Library", "DISABLED") ;
	}

	int startDevice = 0;
	int endDevice = 0;
	int deviceCount;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess) {
		printError(" In Grid_Cuda::info, hipGetDeviceCount", (int) error_id) ;
	}

	if (deviceCount == 0) {
		printError(" No GPU found") ;
	}
	else
	{
		printInfo(MASTER, " Number of GPUs found", deviceCount) ;
	}

	startDevice = 0;
	endDevice = deviceCount - 1;

	for (int currentDevice = startDevice; currentDevice <= endDevice;
			currentDevice++) {
		hipDeviceProp_t deviceProp;
		hipError_t error_id = hipGetDeviceProperties(&deviceProp, currentDevice);

		if (error_id == hipSuccess) {
			string deviceStr = " Device #" + to_string(currentDevice) + "\t";
			printInfo(MASTER, deviceStr, deviceProp.name) ;

			if (deviceProp.computeMode == hipComputeModeProhibited) {
				printError(" Error: device is running in <Compute Mode Prohibited>") ;
			}
		} else {
			printf("hipGetDeviceProperties returned %d\n-> %s\n", (int)error_id,
					hipGetErrorString(error_id));
		}
	}


	printDebug(FULL_DEBUG, "OUT Grid_Cuda::info");
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::write(string file_name)
{
	printDebug(LIGHT_DEBUG, "IN Grid_Cuda::write");

	// each proc write is own file

	if (Config::Instance()->writeGrid)
	{
		// copy grid from device to host
		copyGridDeviceToHost(ALL_POINTS) ;

		Grid_Cuda::write(file_name) ;
	}

	printDebug(LIGHT_DEBUG, "OUT Grid_Cuda::write");
}


//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::FD_D2_N1(Point_type pointType, const Grid& Wgrid, Myint fdOrder)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::FD_D2_N1");

	// check grids are same size
	if (this->sameSize(Wgrid) != true)
	{
		printError("Grid_Cuda::FD_D2_N1, grids have not same size") ;
		return(RTN_CODE_KO) ;
	}

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(INNER_POINTS, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End) ;

	const Myfloat inv_d1  = Myfloat(1.0) / d1 ;
	const Myfloat inv_d2  = Myfloat(1.0) / d2 ;
	const Myfloat inv_d3  = Myfloat(1.0) / d3 ;

	const Myfloat inv2_d1 = inv_d1 * inv_d1 ;
	const Myfloat inv2_d2 = inv_d2 * inv_d2 ;
	const Myfloat inv2_d3 = inv_d3 * inv_d3 ;

	Myfloat * d_w = ((Grid_Cuda&) Wgrid).d_grid_3d ;
	Myfloat * d_u = this->d_grid_3d ;

	dim3 BlkSize(gpuBlkSize1, gpuBlkSize2, gpuBlkSize3) ;
	dim3 GridSize(gpuGridSize1, gpuGridSize2, gpuGridSize3) ;

	if (fdOrder == 2)
	{
		kernel_FD_D2_N1_O2<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 4)
	{
		kernel_FD_D2_N1_O4<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 6)
	{
		kernel_FD_D2_N1_O6<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 8)
	{
		kernel_FD_D2_N1_O8<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 10)
	{
		kernel_FD_D2_N1_O10<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 12)
	{
		kernel_FD_D2_N1_O12<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 14)
	{
		kernel_FD_D2_N1_O14<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 16)
	{
		kernel_FD_D2_N1_O16<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}

	cudaCheckError();
	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::FD_D2_N1");
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::FD_D2_N2(Point_type pointType, const Grid& Wgrid, Myint fdOrder)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::FD_D2_N2");

	// check grids are same size
	if (this->sameSize(Wgrid) != true)
	{
		printError("Grid_Cuda::FD_D2_N2, grids have not same size") ;
		return(RTN_CODE_KO) ;
	}

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(INNER_POINTS, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End) ;

	const Myfloat inv_d1  = Myfloat(1.0) / d1 ;
	const Myfloat inv_d2  = Myfloat(1.0) / d2 ;
	const Myfloat inv_d3  = Myfloat(1.0) / d3 ;

	const Myfloat inv2_d1 = inv_d1 * inv_d1 ;
	const Myfloat inv2_d2 = inv_d2 * inv_d2 ;
	const Myfloat inv2_d3 = inv_d3 * inv_d3 ;

	Myfloat * d_w = ((Grid_Cuda&) Wgrid).d_grid_3d ;
	Myfloat * d_u = this->d_grid_3d ;

	dim3 BlkSize(gpuBlkSize1, gpuBlkSize2, gpuBlkSize3) ;
	dim3 GridSize(gpuGridSize1, gpuGridSize2, gpuGridSize3) ;

	if (fdOrder == 2)
	{
		kernel_FD_D2_N2_O2<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 4)
	{
		kernel_FD_D2_N2_O4<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 6)
	{
		kernel_FD_D2_N2_O6<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 8)
	{
		kernel_FD_D2_N2_O8<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 10)
	{
		kernel_FD_D2_N2_O10<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 12)
	{
		kernel_FD_D2_N2_O12<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 14)
	{
		kernel_FD_D2_N2_O14<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 16)
	{
		kernel_FD_D2_N2_O16<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}

	cudaCheckError();
	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::FD_D2_N2");
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::FD_D2_N3(Point_type pointType, const Grid& Wgrid, Myint fdOrder)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::FD_D2_N3");

	// check grids are same size
	if (this->sameSize(Wgrid) != true)
	{
		printError("Grid_Cuda::FD_D2_N3, grids have not same size") ;
		return(RTN_CODE_KO) ;
	}

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(INNER_POINTS, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End) ;

	const Myfloat inv_d1  = Myfloat(1.0) / d1 ;
	const Myfloat inv_d2  = Myfloat(1.0) / d2 ;
	const Myfloat inv_d3  = Myfloat(1.0) / d3 ;

	const Myfloat inv2_d1 = inv_d1 * inv_d1 ;
	const Myfloat inv2_d2 = inv_d2 * inv_d2 ;
	const Myfloat inv2_d3 = inv_d3 * inv_d3 ;

	Myfloat * d_w = ((Grid_Cuda&) Wgrid).d_grid_3d ;
	Myfloat * d_u = this->d_grid_3d ;

	dim3 BlkSize(gpuBlkSize1, gpuBlkSize2, gpuBlkSize3) ;
	dim3 GridSize(gpuGridSize1, gpuGridSize2, gpuGridSize3) ;

	if (fdOrder == 2)
	{
		kernel_FD_D2_N3_O2<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 4)
	{
		kernel_FD_D2_N3_O4<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 6)
	{
		kernel_FD_D2_N3_O6<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 8)
	{
		kernel_FD_D2_N3_O8<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 10)
	{
		kernel_FD_D2_N3_O10<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 12)
	{
		kernel_FD_D2_N3_O12<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 14)
	{
		kernel_FD_D2_N3_O14<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}
	else if (fdOrder == 16)
	{
		kernel_FD_D2_N3_O16<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
				n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	}

	cudaCheckError();
	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::FD_D2_N3");
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::FD_LAPLACIAN(Point_type pointType, const Grid& Wgrid, Myint fdOrder)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::FD_LAPLACIAN");

	// check grids are same size
	if (this->sameSize(Wgrid) != true)
	{
		printError("Grid_Cuda::FD_LAPLACIAN, grids have not same size") ;
		return(RTN_CODE_KO) ;
	}

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(INNER_POINTS, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End) ;

	const Myfloat inv_d1  = Myfloat(1.0) / d1 ;
	const Myfloat inv_d2  = Myfloat(1.0) / d2 ;
	const Myfloat inv_d3  = Myfloat(1.0) / d3 ;

	const Myfloat inv2_d1 = inv_d1 * inv_d1 ;
	const Myfloat inv2_d2 = inv_d2 * inv_d2 ;
	const Myfloat inv2_d3 = inv_d3 * inv_d3 ;

	Myfloat * d_w = ((Grid_Cuda&) Wgrid).d_grid_3d ;
	Myfloat * d_u = this->d_grid_3d ;

	dim3 BlkSize(gpuBlkSize1, gpuBlkSize2, gpuBlkSize3) ;
	dim3 GridSize(gpuGridSize1, gpuGridSize2, gpuGridSize3) ;

	if (dim == DIM1)
	{
		FD_D2_N1(pointType, Wgrid, fdOrder) ;
	}
	else if (dim == DIM2)
	{
		if (fdOrder == 2)
		{
			kernel_FD_LAPLACIAN_2D_O2<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 4)
		{
			kernel_FD_LAPLACIAN_2D_O4<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 6)
		{
			kernel_FD_LAPLACIAN_2D_O6<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 8)
		{
			kernel_FD_LAPLACIAN_2D_O8<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 10)
		{
			kernel_FD_LAPLACIAN_2D_O10<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 12)
		{
			kernel_FD_LAPLACIAN_2D_O12<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 14)
		{
			kernel_FD_LAPLACIAN_2D_O14<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 16)
		{
			kernel_FD_LAPLACIAN_2D_O16<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
	}
	else if (dim == DIM3)
	{
		if (fdOrder == 2)
		{
			kernel_FD_LAPLACIAN_3D_O2<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 4)
		{
			kernel_FD_LAPLACIAN_3D_O4<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 6)
		{
			kernel_FD_LAPLACIAN_3D_O6<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 8)
		{
			kernel_FD_LAPLACIAN_3D_O8<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 10)
		{
			kernel_FD_LAPLACIAN_3D_O10<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 12)
		{
			kernel_FD_LAPLACIAN_3D_O12<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 14)
		{
			kernel_FD_LAPLACIAN_3D_O14<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 16)
		{
			kernel_FD_LAPLACIAN_3D_O16<<<GridSize, BlkSize>>>(fdOrder, d_w, d_u,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
	}

	cudaCheckError();
	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::FD_LAPLACIAN");
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::computePressureWithFD(Grid& prcGridIn, Grid& coefGridIn, Myint fdOrder)
{

	printDebug(FULL_DEBUG, "In Grid_Cuda::computePressureWithFD") ;

	// check grids are same size
	if (this->sameSize(prcGridIn) != true)
	{
		printError("In Grid_Cuda::computePressureWithFD, grids have not same size") ;
		return(RTN_CODE_KO) ;
	}

	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(INNER_POINTS, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	const Myfloat inv_d1  = Myfloat(1.0) / d1 ;
	const Myfloat inv_d2  = Myfloat(1.0) / d2 ;
	const Myfloat inv_d3  = Myfloat(1.0) / d3 ;

	const Myfloat inv2_d1 = inv_d1 * inv_d1 ;
	const Myfloat inv2_d2 = inv_d2 * inv_d2 ;
	const Myfloat inv2_d3 = inv_d3 * inv_d3 ;

	Myfloat *prc_d_grid_3d = ((Grid_Cuda&) prcGridIn).d_grid_3d ;
	Myfloat *coef_d_grid_3d = ((Grid_Cuda&) coefGridIn).d_grid_3d ;

	dim3 BlkSize(gpuBlkSize1, gpuBlkSize2, gpuBlkSize3) ;
	dim3 GridSize(gpuGridSize1, gpuGridSize2, gpuGridSize3) ;

	if (dim == DIM1)
	{
		if (fdOrder == 2)
		{
			kernel_computePressureWithFD_1D_O2<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 4)
		{
			kernel_computePressureWithFD_1D_O4<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 6)
		{
			kernel_computePressureWithFD_1D_O6<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 8)
		{
			kernel_computePressureWithFD_1D_O8<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 10)
		{
			kernel_computePressureWithFD_1D_O10<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 12)
		{
			kernel_computePressureWithFD_1D_O12<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 14)
		{
			kernel_computePressureWithFD_1D_O14<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 16)
		{
			kernel_computePressureWithFD_1D_O16<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
	}
	else if (dim == DIM2)
	{
		if (fdOrder == 2)
		{
			kernel_computePressureWithFD_2D_O2<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 4)
		{
			kernel_computePressureWithFD_2D_O4<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 6)
		{
			kernel_computePressureWithFD_2D_O6<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 8)
		{
			kernel_computePressureWithFD_2D_O8<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 10)
		{
			kernel_computePressureWithFD_2D_O10<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 12)
		{
			kernel_computePressureWithFD_2D_O12<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 14)
		{
			kernel_computePressureWithFD_2D_O14<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 16)
		{
			kernel_computePressureWithFD_2D_O16<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
	}
	else if (dim == DIM3)
	{
		if (fdOrder == 2)
		{
			kernel_computePressureWithFD_3D_O2<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 4)
		{
			kernel_computePressureWithFD_3D_O4<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 6)
		{
			kernel_computePressureWithFD_3D_O6<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 8)
		{
			kernel_computePressureWithFD_3D_O8<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 10)
		{
			kernel_computePressureWithFD_3D_O10<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 12)
		{
			kernel_computePressureWithFD_3D_O12<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 14)
		{
			kernel_computePressureWithFD_3D_O14<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
		else if (fdOrder == 16)
		{
			kernel_computePressureWithFD_3D_O16<<<GridSize, BlkSize>>>(dim, fdOrder, d_grid_3d, prc_d_grid_3d, coef_d_grid_3d,inv2_d1,inv2_d2,inv2_d3,
					n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
		}
	}

	cudaCheckError();
	hipDeviceSynchronize();

	printDebug(FULL_DEBUG, "Out Grid_Cuda::computePressureWithFD") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::initializeGrid(void)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::initializeGrid") ;

	Grid::initializeGrid() ;

	// for kernels using 3D blocks
	gpuGridSize1 = n1 / gpuBlkSize1 + 1 ;
	gpuGridSize2 = n2 / gpuBlkSize2 + 1 ;
	gpuGridSize3 = n3 / gpuBlkSize3 + 1 ;

	// set device to this MPI rank
	Myint deviceCount;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess) {
		printError(" In Grid_Cuda::initializeGrid, hipGetDeviceCount", (int) error_id) ;
		return(RTN_CODE_KO) ;
	}
	printDebug(FULL_DEBUG, "Device Count", deviceCount) ;

	Myint myDevice = myMpiRank % deviceCount ;
	error_id = hipSetDevice(myDevice);
	if (error_id != hipSuccess) {
		printError(" In Grid_Cuda::initializeGrid, hipSetDevice", (int) error_id) ;
		return(RTN_CODE_KO) ;
	}
	printDebug(FULL_DEBUG, "Device Id" ,myDevice) ;

	if (d_grid_3d == NULL)
	{
		// allocate the grid on the device
		hipMalloc( (void**)&d_grid_3d, npoint * sizeof(Myfloat) );
		cudaCheckError();
	}		

	if (d_help_3d == NULL)
	{
		// allocate 1d array of the device used to perform reduction operation
		hipMalloc( (void**)&d_help_3d, (gpuGridSize) * sizeof(Myfloat) );
		cudaCheckError();
	}

	if (d_help_3d_2 == NULL)
	{
		// allocate 1d array of the device used to perform reduction operation
		hipMalloc( (void**)&d_help_3d_2, (gpuGridSize) * sizeof(Myfloat) );
		cudaCheckError();
	}
	printDebug(FULL_DEBUG, "Out Grid_Cuda::initializeGrid") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------
void Grid_Cuda::fill(Point_type pointType, Myfloat val)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::fill") ;

	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	kernel_fill_const<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d,val,n1,n2,n3,i1Start,i1End,i2Start,i2End,i3Start,i3End);
	hipDeviceSynchronize();
	cudaCheckError();

	printDebug(FULL_DEBUG, "Out Grid_Cuda::fill") ;
}

//-------------------------------------------------------------------------------------------------------
void Grid_Cuda::fill(Point_type pointType, Func_type t1,  Func_type t2, Func_type t3,
		Myfloat64 param1, Myfloat64 param2, Myfloat64 param3, Myfloat64 amp)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::fill") ;

	// this function is critical for validation purpose of hpcscan
	// however, it is not included in the performance benchmark

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End, s1, s2, s3 ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End) ;

	s1 = i1End - i1Start + 1;
	s2 = i2End - i2Start + 1;
	s3 = i3End - i3Start + 1;

	Myfloat64 *val1 = new Myfloat64[s1] ;
	Myfloat64 *val2 = new Myfloat64[s2] ;
	Myfloat64 *val3 = new Myfloat64[s3] ;

	// build the 3 1d functions
	// this is done on the host
#pragma omp simd
	for (Myint64 i1 = i1Start; i1<= i1End; i1++)
	{
		Myfloat64 coord1 = Myfloat64(Orig1 + i1 * d1) ;

		if (dim >= DIM1)
		{
			if (t1 == FUNC_SINE)
			{
				val1[i1-i1Start] = sin(coord1 * param1) ;
			}
			else if (t1 == FUNC_COSINE)
			{
				val1[i1-i1Start] = cos(coord1 * param1) ;
			}
			else if (t1 == FUNC_LINEAR)
			{
				val1[i1-i1Start] = coord1 ;
			}
			else if (t1 == FUNC_CONST)
			{
				val1[i1-i1Start] = 1.0 ;
			}
			else if (t1 == FUNC_RANDOM)
			{
				val1[i1-i1Start] = Myfloat(rand()) / Myfloat(RAND_MAX) ;
			}
			else
			{
				val1[i1-i1Start] = 1.0;
			}
		}
		else
		{
			val1[i1-i1Start] = 1.0 ;
		}
	}

#pragma omp simd
	for (Myint64 i2 = i2Start; i2<= i2End; i2++)
	{
		Myfloat64 coord2 = Myfloat64(Orig2 + i2 * d2) ;
		if (dim >= DIM2)
		{
			if (t2 == FUNC_SINE)
			{
				val2[i2-i2Start] = sin(coord2 * param2) ;
			}
			else if (t2 == FUNC_COSINE)
			{
				val2[i2-i2Start] = cos(coord2 * param2) ;
			}
			else if (t2 == FUNC_LINEAR)
			{
				val2[i2-i2Start] = coord2 ;
			}
			else if (t2 == FUNC_CONST)
			{
				val2[i2-i2Start] = 1.0 ;
			}
			else if (t2 == FUNC_RANDOM)
			{
				val2[i2-i2Start] = Myfloat(rand()) / Myfloat(RAND_MAX) ;
			}
			else
			{
				val2[i2-i2Start] = 1.0;
			}
		}
		else
		{
			val2[i2-i2Start] = 1.0 ;
		}
	}

#pragma omp simd
	for (Myint64 i3 = i3Start; i3<= i3End; i3++)
	{
		Myfloat64 coord3 = Myfloat64(Orig3 + i3 * d3) ;
		if (dim >= DIM3)
		{
			if (t3 == FUNC_SINE)
			{
				val3[i3-i3Start] = sin(coord3 * param3) ;
			}
			else if (t3 == FUNC_COSINE)
			{
				val3[i3-i3Start] = cos(coord3 * param3) ;
			}
			else if (t3 == FUNC_LINEAR)
			{
				val3[i3-i3Start] = coord3 ;
			}
			else if (t3 == FUNC_CONST)
			{
				val3[i3-i3Start] = 1.0 ;
			}
			else if (t3 == FUNC_RANDOM)
			{
				val3[i3-i3Start] = Myfloat(rand()) / Myfloat(RAND_MAX) ;
			}
			else
			{
				val3[i3-i3Start] = 1.0;
			}
		}
		else
		{
			val3[i3-i3Start] = 1.0 ;
		}
	}

	// copy the 1d functions from host to device
	Myfloat64 *d_val1 ;
	hipMalloc( (void**)&d_val1, s1 * sizeof(Myfloat64) );
	hipMemcpy(&(d_val1[0]), &(val1[0]), s1 * sizeof(Myfloat64), hipMemcpyHostToDevice) ;

	Myfloat64 *d_val2 ;
	hipMalloc( (void**)&d_val2, s2 * sizeof(Myfloat64) );
	hipMemcpy(&(d_val2[0]), &(val2[0]), s2 * sizeof(Myfloat64), hipMemcpyHostToDevice) ;

	Myfloat64 *d_val3 ;
	hipMalloc( (void**)&d_val3, s3 * sizeof(Myfloat64) );
	hipMemcpy(&(d_val3[0]), &(val3[0]), s3 * sizeof(Myfloat64), hipMemcpyHostToDevice) ;

	// fill the grid
	kernel_fill_function<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d, d_val1, d_val2, d_val3, amp, n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End);
	hipDeviceSynchronize();
	cudaCheckError();

	delete[] val1 ;
	delete[] val2 ;
	delete[] val3 ;

	hipFree(d_val1);
	hipFree(d_val2);
	hipFree(d_val3);

	printDebug(FULL_DEBUG, "Out Grid_Cuda::fill") ;
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::fillArray(Myfloat val)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::fillArray");

	Myint64 gridSize = n1*n2*n3;
	kernel_fillArray<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d, val, gridSize) ;

	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::fillArray");
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::copyArray(const Grid& gridIn)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::copyArray");

	Myfloat *gridIn_d_grid_3d = ((Grid_Cuda&) gridIn).d_grid_3d ;
	Myint64 gridSize = n1*n2*n3;
	kernel_copyArray<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d, gridIn_d_grid_3d, gridSize) ;

	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::copyArray");
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::addArray(const Grid& gridIn1, const Grid& gridIn2)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::addArray");

	Myfloat *gridIn1_d_grid_3d = ((Grid_Cuda&) gridIn1).d_grid_3d ;
	Myfloat *gridIn2_d_grid_3d = ((Grid_Cuda&) gridIn2).d_grid_3d ;
	Myint64 gridSize = n1*n2*n3;
	kernel_addArray<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d, gridIn1_d_grid_3d, gridIn2_d_grid_3d, gridSize) ;

	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::addArray");
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::multiplyArray(const Grid& gridIn1, const Grid& gridIn2)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::multiplyArray");

	Myfloat *gridIn1_d_grid_3d = ((Grid_Cuda&) gridIn1).d_grid_3d ;
	Myfloat *gridIn2_d_grid_3d = ((Grid_Cuda&) gridIn2).d_grid_3d ;
	Myint64 gridSize = n1*n2*n3;
	kernel_multiplyArray<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d, gridIn1_d_grid_3d, gridIn2_d_grid_3d, gridSize) ;

	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::multiplyArray");
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::addUpdateArray(const Grid& gridIn)
{
	printDebug(MID_DEBUG, "IN Grid_Cuda::addUpdateArray");

	Myfloat *gridIn_d_grid_3d = ((Grid_Cuda&) gridIn).d_grid_3d ;
	Myint64 gridSize = n1*n2*n3;
	kernel_addUpdateArray<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d, gridIn_d_grid_3d, gridSize) ;

	hipDeviceSynchronize();

	printDebug(MID_DEBUG, "OUT Grid_Cuda::addUpdateArray");
}

//-------------------------------------------------------------------------------------------------------

Myfloat Grid_Cuda::getMin(Point_type pointType)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::getMin") ;

	Myfloat val = 0 ;

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	kernel_multiBlk_minval<<<gpuGridSize, gpuBlkSize, gpuBlkSize * sizeof(Myfloat)>>>(d_grid_3d, d_help_3d,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End) ;
	hipDeviceSynchronize();

	kernel_singleBlk_minval<<<1, gpuBlkSize>>>(d_help_3d, gpuGridSize) ;
	hipDeviceSynchronize();

	hipMemcpy(&val, &(d_help_3d[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);
	cudaCheckError();

	printDebug(FULL_DEBUG, "Out Grid_Cuda::getMin") ;

	return val ;
}

//-------------------------------------------------------------------------------------------------------

Myfloat Grid_Cuda::getMax(Point_type pointType)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::getMax") ;

	Myfloat val = 0 ;

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	kernel_multiBlk_maxval<<<gpuGridSize, gpuBlkSize, gpuBlkSize * sizeof(Myfloat)>>>(d_grid_3d, d_help_3d,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End) ;
	hipDeviceSynchronize();

	kernel_singleBlk_maxval<<<1, gpuBlkSize>>>(d_help_3d, gpuGridSize) ;
	hipDeviceSynchronize();

	hipMemcpy(&val, &(d_help_3d[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);
	cudaCheckError();

	printDebug(FULL_DEBUG, "Out Grid_Cuda::getMax") ;

	return val ;
}

//-------------------------------------------------------------------------------------------------------

Myfloat Grid_Cuda::L1Err(Point_type pointType, const Grid& gridIn) const
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::L1Err") ;

	// check grids have same size
	if (!(this->sameSize(gridIn)))
	{
		printError("Grid_Cuda::L1Err, grids have different size") ;
		return(-1.0) ;
	}

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End) ;

	Myfloat *gridIn_d_grid_3d = ((Grid_Cuda&) gridIn).d_grid_3d ;
	kernel_multiBlk_sumAbsAndAbsDiff<<<gpuGridSize, gpuBlkSize, 2 * gpuBlkSize * sizeof(Myfloat)>>>(d_grid_3d, gridIn_d_grid_3d, d_help_3d, d_help_3d_2,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End) ;
	hipDeviceSynchronize();

	Myfloat sum1f = 0 ;
	kernel_singleBlk_sum<<<1, gpuBlkSize>>>(d_help_3d, gpuGridSize) ;
	hipDeviceSynchronize();
	hipMemcpy(&sum1f, &(d_help_3d[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);

	Myfloat sum2f = 0 ;
	kernel_singleBlk_sum<<<1, gpuBlkSize>>>(d_help_3d_2, gpuGridSize) ;
	hipDeviceSynchronize();
	hipMemcpy(&sum2f, &(d_help_3d_2[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);

	Myfloat64 sum1 , sum2 ;
	sum1 = sum1f ;
	sum2 = sum2f ;

	// prevent divide by zero
	if (sum2 < MAX_ERR_FLOAT) sum2 = 1.0 * npoint ;
	Myfloat err = sum1 / sum2 ;

	printDebug(LIGHT_DEBUG, "sum1", sum1) ;
	printDebug(LIGHT_DEBUG, "sum2", sum2) ;
	printDebug(LIGHT_DEBUG, "err", err) ;

	if (std::isnan(err))
	{
		printError("In Grid_Cuda::L1Err, std::isnan(err)") ;
	}

	printDebug(FULL_DEBUG, "Out Grid_Cuda::L1Err") ;
	return(err) ;
}

//-------------------------------------------------------------------------------------------------------

Myfloat Grid_Cuda::allProcL1Err(Point_type pointType, const Grid& gridIn) const
{
	printDebug(LIGHT_DEBUG, "IN Grid_Cuda::allProcL1Err");

	// check grids have same size
	if (!(this->sameSize(gridIn)))
	{
		printError("Grid_Cuda::allProcL1Err, grids have different size") ;
		return(-1.0) ;
	}

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End) ;

	Myfloat *gridIn_d_grid_3d = ((Grid_Cuda&) gridIn).d_grid_3d ;
	kernel_multiBlk_sumAbsAndAbsDiff<<<gpuGridSize, gpuBlkSize, 2 * gpuBlkSize * sizeof(Myfloat)>>>(d_grid_3d, gridIn_d_grid_3d, d_help_3d, d_help_3d_2,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End) ;
	hipDeviceSynchronize();

	Myfloat sum1f = 0 ;
	kernel_singleBlk_sum<<<1, gpuBlkSize>>>(d_help_3d, gpuGridSize) ;
	hipDeviceSynchronize();
	hipMemcpy(&sum1f, &(d_help_3d[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);

	Myfloat sum2f = 0 ;
	kernel_singleBlk_sum<<<1, gpuBlkSize>>>(d_help_3d_2, gpuGridSize) ;
	hipDeviceSynchronize();
	hipMemcpy(&sum2f, &(d_help_3d_2[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);

	Myfloat64 sum1Loc = sum1f ;
	Myfloat64 sum2Loc = sum2f ;
	Myfloat64 sum1 = 0.0 ;
	Myfloat64 sum2 = 0.0 ;

	// MPI reduction
	MPI_Reduce(&sum1Loc, &sum1, 1, MPI_MYFLOAT64, MPI_SUM, 0, MPI_COMM_WORLD);
	MPI_Reduce(&sum2Loc, &sum2, 1, MPI_MYFLOAT64, MPI_SUM, 0, MPI_COMM_WORLD);

	// prevent divide by zero
	if (sum2 == 0.0) sum2 = 1.0 * npoint ;
	Myfloat err = sum1 / sum2 ;

	printDebug(LIGHT_DEBUG, "sum1", sum1) ;
	printDebug(LIGHT_DEBUG, "sum2", sum2) ;
	printDebug(LIGHT_DEBUG, "err", err) ;

	if (std::isnan(err))
	{
		printError("In Grid_Cuda::allProcL1Err, std::isnan(err)") ;
	}

	printDebug(LIGHT_DEBUG, "OUT Grid_Cuda::allProcL1Err");
	return(err) ;
}

//-------------------------------------------------------------------------------------------------------
Rtn_code Grid_Cuda::updatePressure(Point_type pointType, const Grid& prcGrid,
		const Grid& coefGrid, const Grid& laplaGrid)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::updatePressure") ;

	//pointType
	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	Myfloat *prcGrid_d_grid_3d = ((Grid_Cuda&) prcGrid).d_grid_3d ;
	Myfloat *coefGrid_d_grid_3d = ((Grid_Cuda&) coefGrid).d_grid_3d ;
	Myfloat *laplaGrid_d_grid_3d = ((Grid_Cuda&) laplaGrid).d_grid_3d ;
	kernel_updatePressure<<<gpuGridSize, gpuBlkSize>>>(d_grid_3d, prcGrid_d_grid_3d, coefGrid_d_grid_3d, laplaGrid_d_grid_3d,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End);

	hipDeviceSynchronize();

	printDebug(FULL_DEBUG, "Out Grid_Cuda::updatePressure") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::exchangeHalo(MPI_comm_mode_type commMode, Point_type pointType)
{
	printDebug(FULL_DEBUG, "IN Grid_Cuda::exchangeHalo");

	if (commMode == MPI_COMM_MODE_SENDRECV)
	{
		printDebug(FULL_DEBUG, "MPI_COMM_MODE_SENDRECV") ;

		MPI_Status status ;
		Myint rankSend, rankRecv ;
		MPI_Datatype typeSend, typeRecv ;
		Myfloat *bufSend, *bufRecv ;
		Myint64 i1Send, i2Send, i3Send, i1Recv, i2Recv, i3Recv ;

		Point_type bufSendPointType, bufRecvPointType ;

		if (pointType == I1HALO1)
		{
			printDebug(FULL_DEBUG, "I1HALO1") ;
			i1Send = i1InnerEnd - haloWidth + 1 ;
			i2Send = i2InnerStart ;
			i3Send = i3InnerStart ;
			i1Recv = i1Halo1Start ;
			i2Recv = i2InnerStart ;
			i3Recv = i3InnerStart ;			
			typeSend = i1HaloDataType ;
			typeRecv = i1HaloDataType ;
			rankSend = i1ProcIdEnd ;
			rankRecv = i1ProcIdStart ;

			bufRecvPointType = I1HALO1 ;
			bufSendPointType = I1INNERHALO2 ;
		}

		else if (pointType == I1HALO2)
		{
			printDebug(FULL_DEBUG, "I1HALO2") ;
			i1Send = i1InnerStart ;
			i2Send = i2InnerStart ;
			i3Send = i3InnerStart ;
			i1Recv = i1Halo2Start ;
			i2Recv = i2InnerStart ;
			i3Recv = i3InnerStart ;		
			typeSend = i1HaloDataType ;
			typeRecv = i1HaloDataType ;
			rankSend = i1ProcIdStart ;
			rankRecv = i1ProcIdEnd ;

			bufRecvPointType = I1HALO2 ;
			bufSendPointType = I1INNERHALO1 ;
		}

		else if (pointType == I2HALO1)
		{
			printDebug(FULL_DEBUG, "I2HALO1") ;
			i1Send = i1InnerStart ;
			i2Send = i2InnerEnd - haloWidth + 1 ;
			i3Send = i3InnerStart ;
			i1Recv = i1InnerStart ;
			i2Recv = i2Halo1Start ;
			i3Recv = i3InnerStart ;			
			typeSend = i2HaloDataType ;
			typeRecv = i2HaloDataType ;
			rankSend = i2ProcIdEnd ;
			rankRecv = i2ProcIdStart ;

			bufRecvPointType = I2HALO1 ;
			bufSendPointType = I2INNERHALO2 ;
		}

		else if (pointType == I2HALO2)
		{
			printDebug(FULL_DEBUG, "I2HALO2") ;
			i1Send = i1InnerStart ;
			i2Send = i2InnerStart ;
			i3Send = i3InnerStart ;
			i1Recv = i1InnerStart ;
			i2Recv = i2Halo2Start ;
			i3Recv = i3InnerStart ;		
			typeSend = i2HaloDataType ;
			typeRecv = i2HaloDataType ;
			rankSend = i2ProcIdStart ;
			rankRecv = i2ProcIdEnd ;

			bufRecvPointType = I2HALO2 ;
			bufSendPointType = I2INNERHALO1 ;
		}

		else if (pointType == I3HALO1)
		{
			printDebug(FULL_DEBUG, "I3HALO1") ;
			i1Send = i1InnerStart ;
			i2Send = i2InnerStart ;
			i3Send = i3InnerEnd - haloWidth + 1 ;
			i1Recv = i1InnerStart ;
			i2Recv = i2InnerStart ;
			i3Recv = i3Halo1Start ;			
			typeSend = i3HaloDataType ;
			typeRecv = i3HaloDataType ;
			rankSend = i3ProcIdEnd ;
			rankRecv = i3ProcIdStart ;

			bufRecvPointType = I3HALO1 ;
			bufSendPointType = I3INNERHALO2 ;
		}

		else if (pointType == I3HALO2)
		{
			printDebug(FULL_DEBUG, "I3HALO2") ;
			i1Send = i1InnerStart ;
			i2Send = i2InnerStart ;
			i3Send = i3InnerStart ;
			i1Recv = i1InnerStart ;
			i2Recv = i2InnerStart ;
			i3Recv = i3Halo2Start ;		
			typeSend = i3HaloDataType ;
			typeRecv = i3HaloDataType ;
			rankSend = i3ProcIdStart ;
			rankRecv = i3ProcIdEnd ;

			bufRecvPointType = I3HALO2 ;
			bufSendPointType = I3INNERHALO1 ;
		}

		else
		{
			printError("IN Grid_Cuda::exchangeHalo, invalid pointType", pointType) ;
			return(RTN_CODE_KO) ;
		}

		if (!(Config::Instance()->gpuMpiAware))
		{

			Myint64 idxSend = i1Send+i2Send*n1+i3Send*n1*n2 ;
			Myint64 idxRecv = i1Recv+i2Recv*n1+i3Recv*n1*n2 ;
			bufSend = &(grid_3d[idxSend]) ;
			bufRecv = &(grid_3d[idxRecv]) ;

			// copy halo to send from device to host
			if (rankSend != MPI_PROC_NULL)
				copyGridDeviceToHost(bufSendPointType) ;

			// call MPI_Sendrecv
			printDebug(FULL_DEBUG, "MPI_Sendrecv", rankSend, rankRecv) ;
			MPI_Sendrecv(bufSend, 1, typeSend, rankSend, 0,
					bufRecv, 1, typeRecv, rankRecv, 0,
					MPI_COMM_WORLD, &status);

			// copy halo received from host to device
			if (rankRecv != MPI_PROC_NULL)
				copyGridHostToDevice(bufRecvPointType) ;

		}
		else
		{
			Myint64 idxSend = i1Send+i2Send*n1+i3Send*n1*n2 ;
			Myint64 idxRecv = i1Recv+i2Recv*n1+i3Recv*n1*n2 ;
			bufSend = &(d_grid_3d[idxSend]) ;
			bufRecv = &(d_grid_3d[idxRecv]) ;

			// call MPI_Sendrecv
			printDebug(FULL_DEBUG, "MPI_Sendrecv", rankSend, rankRecv) ;
			MPI_Sendrecv(bufSend, 1, typeSend, rankSend, 0,
					bufRecv, 1, typeRecv, rankRecv, 0,
					MPI_COMM_WORLD, &status);
		}

	}
	else
	{
		printError("IN Grid_Cuda::exchangeHalo, invalid commMode", commMode) ;
		return(RTN_CODE_KO) ;
	}

	printDebug(FULL_DEBUG, "OUT Grid_Cuda::exchangeHalo");
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::copyGridDeviceToHost(Point_type pointType)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::copyGridDeviceToHost") ;

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	if (pointType == ALL_POINTS)
	{
		// copy all points with one call to hipMemcpy
		Myint64 idx = 0 ;
		hipMemcpy(&(grid_3d[idx]), &(d_grid_3d[idx]), npoint * sizeof(Myfloat), hipMemcpyDeviceToHost) ;
	}
	else
	{
		for (Myint64 i3 = i3Start; i3<= i3End; i3++)
		{
			for (Myint64 i2 = i2Start; i2<= i2End; i2++)
			{
				// copy 1d segment from i1Start to i1End
				Myint64 i1 = i1Start ;
				Myint64 idx = i1+i2*n1+i3*n1*n2 ;
				Myint64 nn = i1End - i1Start + 1 ;
				hipMemcpy(&(grid_3d[idx]), &(d_grid_3d[idx]), nn * sizeof(Myfloat), hipMemcpyDeviceToHost) ;
			}
		}
	}

	hipDeviceSynchronize();

	printDebug(FULL_DEBUG, "Out Grid_Cuda::copyGridDeviceToHost") ;
}

//-------------------------------------------------------------------------------------------------------

void Grid_Cuda::copyGridHostToDevice(Point_type pointType)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::copyGridHostToDevice") ;

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	if (pointType == ALL_POINTS)
	{
		// copy all points with one call to hipMemcpy
		Myint64 idx = 0 ;
		hipMemcpy(&(d_grid_3d[idx]), &(grid_3d[idx]), npoint * sizeof(Myfloat), hipMemcpyHostToDevice) ;
	}
	else
	{
		for (Myint64 i3 = i3Start; i3<= i3End; i3++)
		{
			for (Myint64 i2 = i2Start; i2<= i2End; i2++)
			{
				// copy 1d segment
				Myint64 i1 = i1Start ;
				Myint64 idx = i1+i2*n1+i3*n1*n2 ;
				Myint64 nn = i1End - i1Start + 1 ;
				hipMemcpy(&(d_grid_3d[idx]), &(grid_3d[idx]), nn * sizeof(Myfloat), hipMemcpyHostToDevice) ;
			}
		}
	}

	hipDeviceSynchronize();

	printDebug(FULL_DEBUG, "Out Grid_Cuda::copyGridHostToDevice") ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::applyBoundaryCondition(BoundCond_type boundCondType)
{
	printDebug(FULL_DEBUG, "In Grid_Cuda::applyBoundaryCondition") ;


	if (boundCondType == NO_BOUND_COND)
	{
		// nothing to do
	}

	else if (boundCondType == BOUND_COND_ANTI_MIRROR)
	{
		Myint64 	i1halo1_i1Start, i1halo1_i1End, i1halo1_i2Start, i1halo1_i2End, i1halo1_i3Start, i1halo1_i3End,
		i1halo2_i1Start, i1halo2_i1End, i1halo2_i2Start, i1halo2_i2End, i1halo2_i3Start, i1halo2_i3End,
		i2halo1_i1Start, i2halo1_i1End, i2halo1_i2Start, i2halo1_i2End, i2halo1_i3Start, i2halo1_i3End,
		i2halo2_i1Start, i2halo2_i1End, i2halo2_i2Start, i2halo2_i2End, i2halo2_i3Start, i2halo2_i3End,
		i3halo1_i1Start, i3halo1_i1End, i3halo1_i2Start, i3halo1_i2End, i3halo1_i3Start, i3halo1_i3End,
		i3halo2_i1Start, i3halo2_i1End, i3halo2_i2Start, i3halo2_i2End, i3halo2_i3Start, i3halo2_i3End;

		getGridIndex(I1HALO1, &i1halo1_i1Start, &i1halo1_i1End, &i1halo1_i2Start, &i1halo1_i2End, &i1halo1_i3Start, &i1halo1_i3End);
		getGridIndex(I1HALO2, &i1halo2_i1Start, &i1halo2_i1End, &i1halo2_i2Start, &i1halo2_i2End, &i1halo2_i3Start, &i1halo2_i3End);
		getGridIndex(I2HALO1, &i2halo1_i1Start, &i2halo1_i1End, &i2halo1_i2Start, &i2halo1_i2End, &i2halo1_i3Start, &i2halo1_i3End);
		getGridIndex(I2HALO2, &i2halo2_i1Start, &i2halo2_i1End, &i2halo2_i2Start, &i2halo2_i2End, &i2halo2_i3Start, &i2halo2_i3End);
		getGridIndex(I3HALO1, &i3halo1_i1Start, &i3halo1_i1End, &i3halo1_i2Start, &i3halo1_i2End, &i3halo1_i3Start, &i3halo1_i3End);
		getGridIndex(I3HALO2, &i3halo2_i1Start, &i3halo2_i1End, &i3halo2_i2Start, &i3halo2_i2End, &i3halo2_i3Start, &i3halo2_i3End);


		kernel_applyBoundaryCondition<<<gpuGridSize, gpuBlkSize>>>(dim, d_grid_3d, n1, n2, n3,
				getNeighbourProc(I1HALO1), i1halo1_i1Start, i1halo1_i1End, i1halo1_i2Start, i1halo1_i2End, i1halo1_i3Start, i1halo1_i3End,
				getNeighbourProc(I1HALO2), i1halo2_i1Start, i1halo2_i1End, i1halo2_i2Start, i1halo2_i2End, i1halo2_i3Start, i1halo2_i3End,
				getNeighbourProc(I2HALO1), i2halo1_i1Start, i2halo1_i1End, i2halo1_i2Start, i2halo1_i2End, i2halo1_i3Start, i2halo1_i3End,
				getNeighbourProc(I2HALO2), i2halo2_i1Start, i2halo2_i1End, i2halo2_i2Start, i2halo2_i2End, i2halo2_i3Start, i2halo2_i3End,
				getNeighbourProc(I3HALO1), i3halo1_i1Start, i3halo1_i1End, i3halo1_i2Start, i3halo1_i2End, i3halo1_i3Start, i3halo1_i3End,
				getNeighbourProc(I3HALO2), i3halo2_i1Start, i3halo2_i1End, i3halo2_i2Start, i3halo2_i2End, i3halo2_i3Start, i3halo2_i3End);

		hipDeviceSynchronize();
	}
	else
	{
		printError("IN Grid_Cuda::applyBoundaryCondition, invalid boundCondType", boundCondType) ;
		return(RTN_CODE_KO) ;
	}


	printDebug(FULL_DEBUG, "Out Grid_Cuda::applyBoundaryCondition") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Myfloat Grid_Cuda::getSumAbs(Point_type pointType) const
{
	printDebug(LIGHT_DEBUG, "IN Grid_Cuda::getSumAbs");

	Myfloat sum = 0 ;

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	kernel_multiBlk_sumAbs<<<gpuGridSize, gpuBlkSize, gpuBlkSize * sizeof(Myfloat)>>>(d_grid_3d, d_help_3d,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End) ;
	hipDeviceSynchronize();

	kernel_singleBlk_sum<<<1, gpuBlkSize>>>(d_help_3d, gpuGridSize) ;
	hipDeviceSynchronize();

	hipMemcpy(&sum, &(d_help_3d[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);
	cudaCheckError();

	// MPI reduction
	Myfloat64 sum2Loc = sum ;
	Myfloat64 sum2 = 0.0 ;
	if (gridType == GRID_LOCAL)
	{
		MPI_Reduce(&sum2Loc, &sum2, 1, MPI_MYFLOAT64, MPI_SUM, 0, MPI_COMM_WORLD);
	}
	else
	{
		sum2 = sum2Loc ;
	}

	printDebug(LIGHT_DEBUG, "sum2", sum2) ;

	if (std::isnan(sum2))
	{
		printError("In Grid_Cuda::getSumAbs, std::isnan(sum2)") ;
	}

	printDebug(LIGHT_DEBUG, "OUT Grid_Cuda::getSumAbs");

	return(sum2) ;
}

//-------------------------------------------------------------------------------------------------------

Myfloat Grid_Cuda::getSumAbsDiff(Point_type pointType, const Grid& gridIn) const
{

	printDebug(LIGHT_DEBUG, "IN Grid_Cuda::getSumAbsDiff");

	// check grids have same size
	if (!(this->sameSize(gridIn)))
	{
		printError("Grid_Cuda::getSumAbsDiff, grids have different size") ;
		return(-1.0) ;
	}

	Myfloat sum = 0 ;

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	Myfloat *gridIn_d_grid_3d = ((Grid_Cuda&) gridIn).d_grid_3d ;
	kernel_multiBlk_sumAbsDiff<<<gpuGridSize, gpuBlkSize, gpuBlkSize * sizeof(Myfloat)>>>(d_grid_3d, gridIn_d_grid_3d, d_help_3d,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End) ;
	hipDeviceSynchronize();

	kernel_singleBlk_sum<<<1, gpuBlkSize>>>(d_help_3d, gpuGridSize) ;
	hipDeviceSynchronize();

	hipMemcpy(&sum, &(d_help_3d[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);
	cudaCheckError();

	// MPI reduction
	Myfloat64 sum1Loc = sum ;
	Myfloat64 sum1 = 0.0 ;
	if (gridType == GRID_LOCAL)
	{
		MPI_Reduce(&sum1Loc, &sum1, 1, MPI_MYFLOAT64, MPI_SUM, 0, MPI_COMM_WORLD);
	}
	else
	{
		sum1 = sum1Loc ;
	}

	printDebug(LIGHT_DEBUG, "sum1", sum1) ;

	if (std::isnan(sum1))
	{
		printError("In Grid_Cuda::getSumAbsDiff, std::isnan(sum1)") ;
	}

	printDebug(LIGHT_DEBUG, "OUT Grid_Cuda::getSumAbsDiff");

	return(sum1) ;
}

//-------------------------------------------------------------------------------------------------------

Myfloat Grid_Cuda::maxErr(Point_type pointType, const Grid& gridIn) const
{
	printDebug(FULL_DEBUG, "IN Grid_Cuda::maxErr");

	// check grids have same size
	if (!(this->sameSize(gridIn)))
	{
		printError("Grid_Cuda::maxErr, grids have different size") ;
		return(-1.0) ;
	}

	Myfloat err = 0 ;

	Myint64 i1Start, i1End, i2Start, i2End, i3Start, i3End ;
	getGridIndex(pointType, &i1Start, &i1End, &i2Start, &i2End, &i3Start, &i3End);

	Myfloat *gridIn_d_grid_3d = ((Grid_Cuda&) gridIn).d_grid_3d ;
	kernel_multiBlk_maxErr<<<gpuGridSize, gpuBlkSize, gpuBlkSize * sizeof(Myfloat)>>>(d_grid_3d, gridIn_d_grid_3d, d_help_3d,
			n1, n2, n3, i1Start, i1End, i2Start, i2End, i3Start, i3End) ;
	hipDeviceSynchronize();

	kernel_singleBlk_maxval<<<1, gpuBlkSize>>>(d_help_3d, gpuGridSize) ;
	hipDeviceSynchronize();

	hipMemcpy(&err, &(d_help_3d[0]), sizeof(Myfloat), hipMemcpyDeviceToHost);
	cudaCheckError();

	printDebug(FULL_DEBUG, "OUT Grid_Cuda::maxErr");
	return(err) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::sendWithMPI(Myint64 nGridPoint, Myint procDestId)
{

	printDebug(FULL_DEBUG, "In Grid_Cuda::sendWithMPI") ;

	if (!Config::Instance()->gpuMpiAware)	
	{
		// copy from device to host
		Myint64 idx = 0 ;
		hipMemcpy(&(grid_3d[idx]), &(d_grid_3d[idx]), nGridPoint * sizeof(Myfloat), hipMemcpyDeviceToHost) ;

		MPI_Send(grid_3d, nGridPoint, MPI_MYFLOAT, procDestId, 0, MPI_COMM_WORLD) ;
	}
	else
	{
		MPI_Send(d_grid_3d, nGridPoint, MPI_MYFLOAT, procDestId, 0, MPI_COMM_WORLD) ;
	}

	printDebug(FULL_DEBUG, "Out Grid_Cuda::sendWithMPI") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::recvWithMPI(Myint64 nGridPoint, Myint procSrcId)
{

	printDebug(FULL_DEBUG, "In Grid_Cuda::recvWithMPI") ;

	MPI_Status status ;

	if (!Config::Instance()->gpuMpiAware)	
	{
		MPI_Recv(grid_3d, nGridPoint, MPI_MYFLOAT, procSrcId, 0, MPI_COMM_WORLD, &status) ;

		// copy from host to device
		Myint64 idx = 0 ;
		hipMemcpy(&(d_grid_3d[idx]), &(grid_3d[idx]), npoint * sizeof(Myfloat), hipMemcpyHostToDevice) ;
	}
	else
	{
		MPI_Recv(d_grid_3d, nGridPoint, MPI_MYFLOAT, procSrcId, 0, MPI_COMM_WORLD, &status) ;
	}

	if (status.MPI_ERROR != MPI_SUCCESS)
	{
		//printError("MPI ERROR", status.MPI_ERROR) ;
	}	

	printDebug(FULL_DEBUG, "Out Grid_Cuda::recvWithMPI") ;
	return(RTN_CODE_OK) ;
}

//-------------------------------------------------------------------------------------------------------

Rtn_code Grid_Cuda::sendRecvWithMPI(const Grid& gridDest, Myint idSend, Myint idRecv, Myint64 nGridPoint)
{

	printDebug(FULL_DEBUG, "In Grid_Cuda::sendRecvWithMPI") ;

	MPI_Status status ;

	if (!Config::Instance()->gpuMpiAware)
	{
		Myfloat *bufSend = grid_3d ;
		Myfloat *bufRecv = gridDest.grid_3d ;

		// copy from device to host
		Myint64 idx = 0 ;
		hipMemcpy(&(grid_3d[idx]), &(d_grid_3d[idx]), nGridPoint * sizeof(Myfloat), hipMemcpyDeviceToHost) ;

		MPI_Sendrecv(bufSend, nGridPoint, MPI_MYFLOAT, idSend, 0,
				bufRecv, nGridPoint, MPI_MYFLOAT, idRecv, 0,
				MPI_COMM_WORLD, &status) ;
		if (status.MPI_ERROR != MPI_SUCCESS)
		{
			//printError("MPI ERROR", status.MPI_ERROR) ;
		}

		// copy from host to device
		Myfloat *gridDest_d_grid_3d = ((Grid_Cuda&) gridDest).d_grid_3d ;
		hipMemcpy(&(gridDest_d_grid_3d[idx]), &(gridDest.grid_3d[idx]), npoint * sizeof(Myfloat), hipMemcpyHostToDevice) ;
	}
	else
	{
		Myfloat *bufSend = d_grid_3d ;
		Myfloat *bufRecv = ((Grid_Cuda&) gridDest).d_grid_3d ;

		MPI_Sendrecv(bufSend, nGridPoint, MPI_MYFLOAT, idSend, 0,
				bufRecv, nGridPoint, MPI_MYFLOAT, idRecv, 0,
				MPI_COMM_WORLD, &status) ;
	}	

	printDebug(FULL_DEBUG, "Out Grid_Cuda::sendRecvWithMPI") ;
	return(RTN_CODE_OK) ;
}

} // namespace hpcscan
